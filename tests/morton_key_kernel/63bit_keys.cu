#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <bitset>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "device/morton.cuh"
#include "kernels/build_sph.cuh"

#include "helper/random.cuh"

int main(int argc, char* argv[])
{
    typedef grace::uinteger64 KeyT;
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(6);
    std::cout.fill('0');

    /*************************************************************/
    /* Compare morton_key_kernel to a CPU loop, for 63-bit keys. */
    /*************************************************************/

    size_t N = 10000;
    bool verbose = false;
    if (argc > 1) {
        N = (size_t)std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        verbose = (std::string(argv[2]) == "true") ? true : false;
    }

    // Generate N random points with double precision co-ordinates in [0, 1).
    thrust::host_vector<double4> h_points(N);
    thrust::transform(thrust::counting_iterator<size_t>(0),
                      thrust::counting_iterator<size_t>(N),
                      h_points.begin(),
                      random_real4_functor<double4>());
    thrust::device_vector<double4> d_points = h_points;

    // Compute keys on host.
    thrust::host_vector<KeyT> h_keys(N);
    const KeyT MAX_KEY = (1u << 21) - 1;
    for (size_t i = 0; i < N; ++i) {
        KeyT ux = static_cast<KeyT>(h_points[i].x * MAX_KEY);
        KeyT uy = static_cast<KeyT>(h_points[i].y * MAX_KEY);
        KeyT uz = static_cast<KeyT>(h_points[i].z * MAX_KEY);

        h_keys[i] = grace::morton::morton_key(ux, uy, uz);
    }

    // Compute keys on device.
    float3 top = make_float3(1., 1., 1.);
    float3 bot = make_float3(0., 0., 0.);
    thrust::device_vector<KeyT> d_keys(N);
    grace::morton_keys_sph(d_points, top, bot, d_keys);

    // Check device keys against host keys.
    int errors = 0;
    thrust::host_vector<KeyT> h_d_keys = d_keys;
    for (size_t i = 0; i < N; ++i)
    {
        KeyT h_key = h_keys[i];
        KeyT d_key = h_d_keys[i];
        if (h_key != d_key)
        {
            ++errors;

            if (!verbose) {
                continue;
            }

            std::cout << "host morton key != device morton key" << std::endl
                      << "(x, y, z): " << " ("
                      << std::setw(8) << h_points[i].x << ", "
                      << std::setw(8) << h_points[i].y << ", "
                      << std::setw(8) << h_points[i].z << ")" << std::endl
                      << "Host key:   " << std::bitset<64>(h_key)
                      << std::endl
                      << "Device key: " << std::bitset<64>(d_key)
                      << std::endl
                      << "Diff bits:  " << std::bitset<64>(h_key ^ d_key)
                      << std::endl << std::endl;
        }
    }

    if (errors != 0 && verbose) {
        std::cout << std::endl;
    }

    if (errors == 0) {
        std::cout << "PASSED" << std::endl;
    }
    else {
        std::cout << errors << " of " << N << " keys did not match host"
                  << std::endl
                  << "FAILED" << std::endl;
    }

    return errors == 0 ? EXIT_SUCCESS : EXIT_FAILURE;
}
