#include "hip/hip_runtime.h"
// Due to a bug in thrust, this must appear before thrust/sort.h
// The simplest solution is to put it here, despite already being included in
// all of the includes which require it.
// See http://stackoverflow.com/questions/23352122
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <sstream>
#include <fstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include "../kernel_config.h"
#include "../nodes.h"
#include "../ray.h"
#include "../utils.cuh"
#include "../kernels/bintree_build.cuh"
#include "../kernels/bintree_trace.cuh"
#include "../kernels/gen_rays.cuh"
#include "../kernels/morton.cuh"
#include "../kernels/sort.cuh"


int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;

    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(3);


    /* Initialize run parameters. */

    unsigned int device_ID = 0;
    unsigned int N_rays = 145000;
    unsigned int max_per_leaf = 100;
    unsigned int N_iter = 2;

    if (argc > 1) {
        device_ID = (unsigned int) std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        N_rays = (unsigned int) std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        max_per_leaf = (unsigned int) std::strtol(argv[3], NULL, 10);
    }
    if (argc > 4) {
        N_iter = (unsigned int) std::strtol(argv[4], NULL, 10);
    }


    /* Read in Gadget file. */

    std::ifstream file;
    std::string fname = "Data_025";

    // Arrays are resized in read_gadget_gas().
    thrust::host_vector<float4> h_spheres_xyzr(1);
    thrust::host_vector<unsigned int> h_gadget_IDs(1);
    thrust::host_vector<float> h_masses(1);
    thrust::host_vector<float> h_rho(1);

    file.open(fname.c_str(), std::ios::binary);
    grace::read_gadget_gas(file, h_spheres_xyzr,
                                 h_gadget_IDs,
                                 h_masses,
                                 h_rho);
    file.close();

    size_t N = h_spheres_xyzr.size();

    // Gadget IDs and masses unused.
    h_gadget_IDs.clear(); h_gadget_IDs.shrink_to_fit();
    h_masses.clear(); h_masses.shrink_to_fit();


    /* Output run parameters and device properties to console. */

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);

    std::cout << "Device " << device_ID
                    << ":                   " << deviceProp.name << std::endl;
    std::cout << "TRACE_THREADS_PER_BLOCK:    " << TRACE_THREADS_PER_BLOCK
            << std::endl;
    std::cout << "MAX_BLOCKS:                 " << MAX_BLOCKS << std::endl;
    std::cout << "Gadget data file:           " << fname << std::endl;
    std::cout << "Number of gas particles:    " << N << std::endl;
    std::cout << "Number of rays:             " << N_rays << std::endl;
    std::cout << "Max particles per leaf:     " << max_per_leaf << std::endl;
    std::cout << "Number of iterations:       " << N_iter << std::endl;
    std::cout << std::endl << std::endl;


{ // Device code.


    /* Build the tree. */

    thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;
    thrust::device_vector<float> d_rho = h_rho;
    thrust::device_vector<grace::uinteger32> d_keys(N);

    grace::morton_keys(d_keys, d_spheres_xyzr);
    grace::sort_by_key(d_keys, d_spheres_xyzr, d_rho);

    grace::Nodes d_nodes(N-1);
    grace::Leaves d_leaves(N);

    grace::build_nodes(d_nodes, d_leaves, d_keys, max_per_leaf);
    grace::compact_nodes(d_nodes, d_leaves);
    grace::find_AABBs(d_nodes, d_leaves, d_spheres_xyzr);


    /* Compute information needed for ray generation; rays are emitted from the
     * box centre and of sufficient length to be terminated outside the box.
     */

    // Assume x, y and z spatial extents are similar.
    float min, max;
    grace::min_max_x(&min, &max, d_spheres_xyzr);
    float x_centre = (max + min) / 2.;
    float y_centre = x_centre;
    float z_centre = x_centre;
    float length = 2 * (max - min) * sqrt(3);


    /* Profile the tracing performance by tracing rays through the simulation
     * data and i) accumulating density and ii) saving column densities
     * and distances to each intersected particle.  Repeat N_iter times.
     */

    hipEvent_t part_start, part_stop;
    hipEvent_t tot_start, tot_stop;
    float elapsed;
    double gen_ray_tot, sort_rho_dists_tot;
    double trace_rho_tot, trace_full_tot;
    double all_tot;
    hipEventCreate(&part_start);
    hipEventCreate(&part_stop);
    hipEventCreate(&tot_start);
    hipEventCreate(&tot_stop);

    for (int i=0; i<N_iter; i++) {
        hipEventRecord(tot_start);

        thrust::device_vector<grace::Ray> d_rays(N_rays);
        thrust::device_vector<float> d_traced_rho(N_rays);

        hipEventRecord(part_start);
        grace::uniform_random_rays(d_rays,
                                   x_centre, y_centre, z_centre, length);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        gen_ray_tot += elapsed;

        hipEventRecord(part_start);
        grace::trace_property<float>(d_rays,
                                     d_traced_rho,
                                     d_nodes,
                                     d_leaves,
                                     d_spheres_xyzr,
                                     d_rho);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        trace_rho_tot += elapsed;


        /* Full trace. */

        // Indices of particles for all ray-particle intersections.
        thrust::device_vector<unsigned int> d_hit_indices;
        // Distances, from the ray origin, to all ray-particle intersections.
        thrust::device_vector<float> d_hit_distances;
        // Offsets into the above vector where each ray's data starts.
        thrust::device_vector<unsigned int> d_ray_offsets(N_rays);

        hipEventRecord(part_start);
        grace::trace<float>(d_rays,
                            d_traced_rho,
                            d_ray_offsets,
                            d_hit_indices,
                            d_hit_distances,
                            d_nodes,
                            d_leaves,
                            d_spheres_xyzr,
                            d_rho); // For RT, we'd pass ~number counts.
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        trace_full_tot += elapsed;

        // If offets = [0, 3, 3, 7], then
        //    segments = [0, 0, 0, 1, 1, 1, 1, 2(, 2 ... )]
        thrust::device_vector<unsigned int> d_ray_segments(d_hit_indices.size());

        hipEventRecord(part_start);
        grace::offsets_to_segments(d_ray_offsets, d_ray_segments);
        grace::sort_by_distance(d_hit_distances,
                                d_ray_segments,
                                d_hit_indices,
                                d_traced_rho);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        sort_rho_dists_tot += elapsed;

        /* End of full trace. */

        hipEventRecord(tot_stop);
        hipEventSynchronize(tot_stop);
        hipEventElapsedTime(&elapsed, tot_start, tot_stop);
        all_tot += elapsed;


        // Must be done in-loop for hipMemGetInfo to return relevant results.
        if (i == 0) {
            float trace_bytes = 0.0;
            float unused_bytes = 0.0;
            trace_bytes += d_rays.size() * sizeof(grace::Ray);
            trace_bytes += d_traced_rho.size() * sizeof(float);
            trace_bytes += d_ray_offsets.size() * sizeof(float);
            trace_bytes += d_hit_indices.size() * sizeof(unsigned int);
            trace_bytes += d_nodes.hierarchy.size() * sizeof(int4);
            trace_bytes += d_nodes.AABB.size() * 3*sizeof(float4);
            trace_bytes += d_leaves.indices.size() * sizeof(int4);
            trace_bytes += d_spheres_xyzr.size() * sizeof(float4);
            trace_bytes += d_rho.size() * sizeof(float);
            // Integral lookup.
            trace_bytes += grace::N_table * sizeof(float);
            trace_bytes += d_ray_segments.size() * sizeof(unsigned int);

            unused_bytes += d_keys.size() * sizeof(unsigned int);
            unused_bytes += d_nodes.level.size() * sizeof(unsigned int);
            // Ray keys, used when generating rays.
            unused_bytes += d_rays.size() * sizeof(unsigned int);

            std::cout << "Total memory for full trace kernel:        "
                      << trace_bytes / (1024.*1024.*1024.) << " GiB"
                      << std::endl;
            std::cout << "Allocated memory not used in trace kernel: "
                      << unused_bytes / (1024.*1024.*1024.) << " GiB"
                      << std::endl;

            size_t avail, total;
            hipMemGetInfo(&avail, &total);
            std::cout << "Free memory:  " << avail / (1024.*1024.*1024.)
                      << " GiB" << std::endl;
            std::cout << "Total memory: " << total / (1024.*1024.*1024.)
                      << " GiB" << std::endl;
            std::cout << std::endl;
        }
    }

    std::cout << "Time for generating and sorting rays:   ";
    std::cout.width(8);
    std::cout << gen_ray_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for cummulative density tracing:   ";
    std::cout.width(8);
    std::cout << trace_rho_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for full tracing:                  ";
    std::cout.width(8);
    std::cout << trace_full_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for sort-by-distance:              ";
    std::cout.width(8);
    std::cout << sort_rho_dists_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for total (inc. memory ops):       ";
    std::cout.width(8);
    std::cout << all_tot / N_iter << " ms" << std::endl;

    std::cout << std::endl;

} // End device code.

    // Exit cleanly to ensure full profiler trace.
    hipDeviceReset();
    return 0;
}
