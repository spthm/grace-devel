#include "hip/hip_runtime.h"
#include <cmath>
#include <sstream>
#include <fstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "utils.cuh"
#include "../types.h"
#include "../nodes.h"
#include "../ray.h"
#include "../kernels/morton.cuh"
#include "../kernels/bintree_build.cuh"
#include "../kernels/bintree_trace.cuh"

int main(int argc, char* argv[])
{

    unsigned int N_rays = 250000;
    if (argc > 1) {
        N_rays = (unsigned int) std::strtol(argv[1], NULL, 10);
    }
    float N_rays_side = floor(pow(N_rays, 0.500001));

    std::ifstream file;
    std::string fname = "Data_025";
    std::cout << "Reading in data from Gadget file " << fname << "..."
              << std::endl;

    // Read in gas data from Gadget-2 file.
    // Arrays are resized in read_gadget_gas().
    thrust::host_vector<float4> h_spheres_xyzr(1);
    thrust::host_vector<float> h_masses(1);
    thrust::host_vector<float> h_rho(1);

    file.open(fname.c_str(), std::ios::binary);
    grace::read_gadget_gas(file, h_spheres_xyzr, h_masses, h_rho);
    file.close();

    size_t N = h_spheres_xyzr.size();
    std::cout << "Will trace " << N_rays << " rays through " << N
              << " particles..." << std::endl;
    std::cout << std::endl;

    // Masses unused.
    h_masses.clear(); h_masses.shrink_to_fit();

// Device code.
{
    thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;
    thrust::device_vector<float> d_rho = h_rho;

    // Set the tree AABB.
    float min_x, max_x;
    grace::min_max_x(&min_x, &max_x, d_spheres_xyzr);

    float min_y, max_y;
    grace::min_max_y(&min_y, &max_y, d_spheres_xyzr);

    float min_z, max_z;
    grace::min_max_z(&min_z, &max_z, d_spheres_xyzr);

    float min_r, max_r;
    grace::min_max_w(&min_r, &max_r, d_spheres_xyzr);

    float4 bot = make_float4(min_x, min_y, min_z, 0.f);
    float4 top = make_float4(max_x, max_y, max_z, 0.f);

    // Generate morton keys based on particles' positions.
    thrust::device_vector<grace::uinteger32> d_keys(N);
    thrust::device_vector<grace::uinteger32> d_keys_2(N);
    grace::morton_keys(d_spheres_xyzr, d_keys, bot, top);
    thrust::copy(d_keys.begin(), d_keys.end(), d_keys_2.begin());

    // Sort particle positions and smoothing lengths by their keys.
    thrust::sort_by_key(d_keys_2.begin(), d_keys_2.end(),
                        d_spheres_xyzr.begin());
    d_keys_2.clear(); d_keys_2.shrink_to_fit();
    // Sort other properties by the same keys.
    thrust::device_vector<int> d_indices(N);
    thrust::device_vector<float> d_sorted(N);
    thrust::sequence(d_indices.begin(), d_indices.end());
    thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

    thrust::gather(d_indices.begin(), d_indices.end(),
                   d_rho.begin(), d_sorted.begin());
    d_rho = d_sorted;

    // Clear temporary storage.
    d_sorted.clear(); d_sorted.shrink_to_fit();
    d_indices.clear(); d_indices.shrink_to_fit();

    // Build the tree hierarchy from the keys.
    grace::Nodes d_nodes(N-1);
    grace::Leaves d_leaves(N);
    grace::build_nodes(d_nodes, d_leaves, d_keys);
    // Keys no longer needed.
    d_keys.clear(); d_keys.shrink_to_fit();
    grace::find_AABBs(d_nodes, d_leaves, d_spheres_xyzr);

    // Generate the rays, emitted from box side (X, Y, min_z-max_r) and of
    // length (max_z + max_r) - (min_z - max_r).
    // For simplicity, the ray (ox, oy) limits are determined only by the
    // particle min(x, y) / max(x, y) limits; smoothing lengths are ignored.
    // This ensures that rays at the edge will hit something!
    float span_x = max_x - min_x;
    float span_y = max_y - min_y;
    float span_z = 2*max_r + max_z - min_z;
    float spacer_x = span_x / N_rays_side;
    float spacer_y = span_y / N_rays_side;
    thrust::host_vector<grace::Ray> h_rays(N_rays);
    thrust::host_vector<grace::uinteger32> h_keys(N_rays);
    int i, j;
    float ox, oy;
    for (i=0, ox=min_x; i<N_rays_side; ox+=spacer_x, i++)
    {
        for (j=0, oy=min_y; j<N_rays_side; oy+=spacer_y, j++)
        {
            // All rays point in +ve z direction.
            h_rays[i*N_rays_side + j].dx = 0.0f;
            h_rays[i*N_rays_side + j].dy = 0.0f;
            h_rays[i*N_rays_side + j].dz = 1.0f;

            h_rays[i*N_rays_side + j].ox = ox;
            h_rays[i*N_rays_side + j].oy = oy;
            h_rays[i*N_rays_side + j].oz = min_z - max_r;

            h_rays[i*N_rays_side + j].length = span_z;
            h_rays[i*N_rays_side + j].dclass = 7;

            // Since all rays are PPP, base key on origin instead.
            // morton_key(float, float, float) requires floats in (0, 1).
            h_keys[i*N_rays_side + j] = grace::morton_key((ox-min_x)/span_x,
                                                          (oy-min_y)/span_y,
                                                          0.0f);
        }
    }

    // Sort rays by Morton key and trace for per-ray hit couynts.
    thrust::sort_by_key(h_keys.begin(), h_keys.end(), h_rays.begin());
    thrust::device_vector<grace::Ray> d_rays = h_rays;
    thrust::device_vector<unsigned int> d_hit_counts(N_rays);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    grace::gpu::trace_hitcount<<<28, TRACE_THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(d_rays.data()),
        d_rays.size(),
        thrust::raw_pointer_cast(d_hit_counts.data()),
        thrust::raw_pointer_cast(d_nodes.left.data()),
        thrust::raw_pointer_cast(d_nodes.right.data()),
        thrust::raw_pointer_cast(d_nodes.AABB.data()),
        d_nodes.left.size(),
        thrust::raw_pointer_cast(d_spheres_xyzr.data()));
    CUDA_HANDLE_ERR( hipPeekAtLastError() );
    CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    int max_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                  0, thrust::maximum<unsigned int>());
    int min_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                  N+1, thrust::minimum<unsigned int>());
    float mean_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                     0, thrust::plus<unsigned int>())
                                    / float(N_rays);
    std::cout << "Time for hit-count tracing kernel: " << elapsed << " ms"
              << std::endl;


    thrust::device_vector<float> d_traced_rho(N_rays);
    // Copy tabulated kernel integrals to device.
    thrust::device_vector<float> d_b_integrals(grace::kernel_integral_table,
                                               grace::kernel_integral_table+51);
    // Trace and integrate through smoothing kernels, accumulating density.
    hipEventRecord(start);
    grace::gpu::trace_property<<<28, TRACE_THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(d_rays.data()),
        d_rays.size(),
        thrust::raw_pointer_cast(d_traced_rho.data()),
        thrust::raw_pointer_cast(d_nodes.left.data()),
        thrust::raw_pointer_cast(d_nodes.right.data()),
        thrust::raw_pointer_cast(d_nodes.AABB.data()),
        d_nodes.left.size(),
        thrust::raw_pointer_cast(d_spheres_xyzr.data()),
        thrust::raw_pointer_cast(d_rho.data()),
        thrust::raw_pointer_cast(d_b_integrals.data()));
    CUDA_HANDLE_ERR( hipPeekAtLastError() );
    CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    // Find min, max of output vector for 'plotting'.
    float max_rho = thrust::reduce(d_traced_rho.begin(), d_traced_rho.end(),
                                   0.0f, thrust::maximum<float>());
    float min_rho = thrust::reduce(d_traced_rho.begin(), d_traced_rho.end(),
                                   1E20, thrust::minimum<float>());
    float mean_rho = thrust::reduce(d_traced_rho.begin(), d_traced_rho.end(),
                                    0.0f, thrust::plus<float>()) / d_traced_rho.size();
    std::cout << "Time for acummulating integrating tracing kernel: "
              << elapsed << " ms" << std::endl;


    // Allocate output array based on per-ray hit counts, and calculate
    // individual ray offsets into this array.
    // int last_ray_hits = d_hit_counts[N_rays-1];
    // thrust::exclusive_scan(d_hit_counts.begin(), d_hit_counts.end(),
    //                        d_hit_counts.begin());
    // thrust::device_vector<float> d_trace_output(d_hit_counts[N_rays-1]+
    //                                             last_ray_hits);
    // thrust::device_vector<float> d_trace_distances(d_trace_output.size());

    // // Trace and integrate through smoothing kernels, accumulating density.
    // hipEventRecord(start);
    // grace::gpu::trace<<<28, TRACE_THREADS_PER_BLOCK>>>(
    //     thrust::raw_pointer_cast(d_rays.data()),
    //     d_rays.size(),
    //     thrust::raw_pointer_cast(d_trace_output.data()),
    //     thrust::raw_pointer_cast(d_trace_distances.data()),
    //     thrust::raw_pointer_cast(d_hit_counts.data()),
    //     thrust::raw_pointer_cast(d_nodes.data()),
    //     thrust::raw_pointer_cast(d_leaves.data()),
    //     d_nodes.size(),
    //     thrust::raw_pointer_cast(d_spheres_xyzr.data()),
    //     thrust::raw_pointer_cast(d_rho.data()),
    //     thrust::raw_pointer_cast(d_b_integrals.data()));
    // CUDA_HANDLE_ERR( hipPeekAtLastError() );
    // CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsed, start, stop);
    // std::cout << "Time for per-intersection integrating kernel: " << elapsed
    //           << " ms" << std::endl;

    // // Sort output arrays based on hit distances.
    // thrust::host_vector<int> h_hit_counts = d_hit_counts;
    // double t = 0.0;
    // for (int i=0; i<N_rays_side; i++) {
    //     int r_start = h_hit_counts[i];
    //     int r_end;
    //     if (i == N_rays-1)
    //         r_end = h_hit_counts[i] + last_ray_hits - 1;
    //     else
    //         r_end = h_hit_counts[i+1] - 1;
    //     hipEventRecord(start);
    //     thrust::sort_by_key(d_trace_distances.begin()+r_start,
    //                         d_trace_distances.begin()+r_end,
    //                         d_trace_output.begin()+r_start);
    //     hipEventRecord(stop);
    //     hipEventSynchronize(stop);
    //     hipEventElapsedTime(&elapsed, start, stop);
    //     t += elapsed;
    // }
    // std::cout << "Time for per-intersection sorting loop: " << t << " ms"
    //           << std::endl;
    std::cout << std::endl;

    std::cout << "Number of rays:       " << N_rays << std::endl;
    std::cout << "Number of particles:  " << N << std::endl;
    std::cout << "Mean hits:            " << mean_hits << std::endl;
    std::cout << "Max hits:             " << max_hits << std::endl;
    std::cout << "Min hits:             " << min_hits << std::endl;
    std::cout << "Mean output           " << mean_rho << std::endl;
    std::cout << "Max output:           " << max_rho << std::endl;
    std::cout << "Min output:           " << min_rho << std::endl;
    std::cout << std::endl;

    // Sort ray hit and ray data such that increasing the index moves us along
    // x first, then y.
    thrust::host_vector<float> h_pos_keys(N_rays);
    for (int i=0; i<N_rays; i++) {
        h_pos_keys[i] = h_rays[i].ox + (2*span_x)*h_rays[i].oy;
    }
    thrust::host_vector<float> h_traced_rho = d_traced_rho;
    thrust::host_vector<int> h_indices(N_rays);
    thrust::sequence(h_indices.begin(), h_indices.end());
    thrust::sort_by_key(h_pos_keys.begin(), h_pos_keys.end(),
                        h_indices.begin());
    {
        thrust::host_vector<float> h_sorted(N_rays);
        thrust::gather(h_indices.begin(), h_indices.end(),
                       h_traced_rho.begin(), h_sorted.begin());
        h_traced_rho = h_sorted;
    }
    {
        thrust::host_vector<grace::Ray> h_sorted(N_rays);
        thrust::gather(h_indices.begin(), h_indices.end(),
                       h_rays.begin(), h_sorted.begin());
        h_rays = h_sorted;
    }
    h_indices.clear(); h_indices.shrink_to_fit();

    // Use log10(rho) because of the increased dynamic range.
    for (int i=0; i<N_rays; i++) {
        h_traced_rho[i] = log10(h_traced_rho[i]);
    }
    min_rho = log10(min_rho);
    max_rho = log10(max_rho);

    // See http://stackoverflow.com/questions/2654480
    FILE *f;
    unsigned char *img = NULL;
    int w=N_rays_side;
    int h=N_rays_side;
    int filesize = 54 + 3*w*h;  //w is your image width, h is image height, both int
    if( img )
        free( img );
    img = (unsigned char *)malloc(3*w*h);
    memset(img,0,sizeof(img));

    int r, g, b, x, y;
    float r_max = 150.0f;
    float g_max = 210.0f;
    float b_max = 255.0f;
    for(int i=0; i<w; i++)
    {
        for(int j=0; j<h; j++)
    {
        x=i; y=(h-1)-j;
        r = (int) ( (h_traced_rho[i+w*j] - min_rho)*r_max/(max_rho-min_rho) );
        g = (int) ( (h_traced_rho[i+w*j] - min_rho)*g_max/(max_rho-min_rho) );
        b = (int) ( (h_traced_rho[i+w*j] - min_rho)*b_max/(max_rho-min_rho) );
        if (r > r_max) r=r_max;
        if (g > g_max) g=g_max;
        if (b > b_max) b=b_max;
        img[(x+y*w)*3+2] = (unsigned char)(r);
        img[(x+y*w)*3+1] = (unsigned char)(g);
        img[(x+y*w)*3+0] = (unsigned char)(b);
    }
    }

    unsigned char bmpfileheader[14] = {'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0};
    unsigned char bmpinfoheader[40] = {40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0};
    unsigned char bmppad[3] = {0,0,0};

    bmpfileheader[2] = (unsigned char)(filesize);
    bmpfileheader[3] = (unsigned char)(filesize>>8);
    bmpfileheader[4] = (unsigned char)(filesize>>16);
    bmpfileheader[5] = (unsigned char)(filesize>>24);

    bmpinfoheader[4] =  (unsigned char)(w);
    bmpinfoheader[5] =  (unsigned char)(w>>8);
    bmpinfoheader[6] =  (unsigned char)(w>>16);
    bmpinfoheader[7] =  (unsigned char)(w>>24);
    bmpinfoheader[8] =  (unsigned char)(h);
    bmpinfoheader[9] =  (unsigned char)(h>>8);
    bmpinfoheader[10] = (unsigned char)(h>>16);
    bmpinfoheader[11] = (unsigned char)(h>>24);

    f = fopen("density.bmp","wb");
    fwrite(bmpfileheader,1,14,f);
    fwrite(bmpinfoheader,1,40,f);
    for(i=0; i<h; i++)
    {
        fwrite(img+(w*(h-i-1)*3),3,w,f);
        fwrite(bmppad,1,(4-(w*3)%4)%4,f);
    }
    fclose(f);
} // End device code.  Call all thrust destructors etc. before hipDeviceReset().

    // Exit cleanly to ensure full profiler trace.
    hipDeviceReset();
    return 0;
}
