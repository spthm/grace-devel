#include "hip/hip_runtime.h"
// Due to a bug in thrust, this must appear before thrust/sort.h
// The simplest solution is to put it here, despite already being included in
// all of the includes which require it.
// See http://stackoverflow.com/questions/23352122
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <sstream>
#include <fstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#include "../kernel_config.h"
#include "../nodes.h"
#include "../ray.h"
#include "../utils.cuh"
#include "../kernels/bintree_build.cuh"
#include "../kernels/bintree_trace.cuh"
#include "../kernels/gen_rays.cuh"
#include "../kernels/morton.cuh"
#include "../kernels/sort.cuh"


int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;

    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(3);


    /* Initialize run parameters. */

    unsigned int device_ID = 0;
    unsigned int N_rays = 4532 * 32; // = 145024 ~ max on an M2090 for data set.
    unsigned int max_per_leaf = 32;
    unsigned int N_iter = 2;

    if (argc > 1) {
        device_ID = (unsigned int) std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        N_rays = 32 * (unsigned int) std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        max_per_leaf = (unsigned int) std::strtol(argv[3], NULL, 10);
    }
    if (argc > 4) {
        N_iter = (unsigned int) std::strtol(argv[4], NULL, 10);
    }


    /* Read in Gadget file. */

    std::ifstream file;
    std::string fname = "Data_025";

    // Arrays are resized in read_gadget_gas().
    thrust::host_vector<float4> h_spheres_xyzr(1);
    thrust::host_vector<unsigned int> h_gadget_IDs(1);
    thrust::host_vector<float> h_masses(1);
    thrust::host_vector<float> h_rho(1);

    file.open(fname.c_str(), std::ios::binary);
    grace::read_gadget_gas(file, h_spheres_xyzr,
                                 h_gadget_IDs,
                                 h_masses,
                                 h_rho);
    file.close();

    size_t N = h_spheres_xyzr.size();

    // Gadget IDs and masses unused.
    h_gadget_IDs.clear(); h_gadget_IDs.shrink_to_fit();
    h_masses.clear(); h_masses.shrink_to_fit();


    /* Output run parameters and device properties to console. */

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);

    std::cout << "Device " << device_ID
                    << ":                   " << deviceProp.name << std::endl;
    std::cout << "TRACE_THREADS_PER_BLOCK:    "
              << grace::TRACE_THREADS_PER_BLOCK << std::endl;
    std::cout << "MAX_BLOCKS:                 "
              << grace::MAX_BLOCKS << std::endl;
    std::cout << "Gadget data file:           " << fname << std::endl;
    std::cout << "Number of gas particles:    " << N << std::endl;
    std::cout << "Number of rays:             " << N_rays << std::endl;
    std::cout << "Max particles per leaf:     " << max_per_leaf << std::endl;
    std::cout << "Number of iterations:       " << N_iter << std::endl;
    std::cout << std::endl << std::endl;


{ // Device code.


    /* Build the tree. */

    thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;
    thrust::device_vector<float> d_rho = h_rho;
    thrust::device_vector<grace::uinteger32> d_keys(N);
    thrust::device_vector<float> d_deltas(N+1);

    grace::morton_keys(d_keys, d_spheres_xyzr);
    grace::sort_by_key(d_keys, d_spheres_xyzr, d_rho);

    grace::Tree d_tree(N, max_per_leaf);

    grace::compute_deltas(d_spheres_xyzr, d_deltas);
    grace::build_tree(d_tree, d_spheres_xyzr, d_deltas, d_spheres_xyzr);


    /* Compute information needed for ray generation; rays are emitted from the
     * box centre and of sufficient length to be terminated outside the box.
     */

    // Assume x, y and z spatial extents are similar.
    float min, max;
    grace::min_max_x(&min, &max, d_spheres_xyzr);
    float x_centre = (max + min) / 2.;
    float y_centre = x_centre;
    float z_centre = x_centre;
    float length = 2 * (max - min) * sqrt(3);


    /* Profile the tracing performance by tracing rays through the simulation
     * data and i) accumulating density and ii) saving column densities
     * and distances to each intersected particle.  Repeat N_iter times.
     */

    hipEvent_t part_start, part_stop;
    hipEvent_t tot_start, tot_stop;
    float elapsed;
    double gen_ray_tot, sort_rho_dists_tot;
    double trace_rho_tot, trace_full_tot, trace_hit_tot;
    double all_tot;
    hipEventCreate(&part_start);
    hipEventCreate(&part_stop);
    hipEventCreate(&tot_start);
    hipEventCreate(&tot_stop);

    for (int i=0; i<N_iter; i++) {
        hipEventRecord(tot_start);

        thrust::device_vector<grace::Ray> d_rays(N_rays);
        hipEventRecord(part_start);
        grace::uniform_random_rays(d_rays,
                                   x_centre, y_centre, z_centre, length);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        gen_ray_tot += elapsed;

        // The cumulated, integrated density for each ray.
        thrust::device_vector<float> d_traced_rho(N_rays);

        hipEventRecord(part_start);
        grace::trace_property<float>(d_rays,
                                     d_traced_rho,
                                     d_tree,
                                     d_spheres_xyzr,
                                     d_rho);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        trace_rho_tot += elapsed;


        /* Full trace. */

        // Indices of particles for all ray-particle intersections.
        thrust::device_vector<unsigned int> d_hit_indices;
        // Distances, from the ray origin, to all ray-particle intersections.
        thrust::device_vector<float> d_hit_distances;
        // Offsets into the above vector where each ray's data starts.
        thrust::device_vector<int> d_ray_offsets(N_rays);

        hipEventRecord(part_start);
        grace::trace<float>(d_rays,
                            d_traced_rho,
                            d_ray_offsets,
                            d_hit_indices,
                            d_hit_distances,
                            d_tree,
                            d_spheres_xyzr,
                            d_rho); // For RT, we'd pass ~number counts.
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        trace_full_tot += elapsed;

        hipEventRecord(part_start);
        grace::sort_by_distance(d_hit_distances,
                                d_ray_offsets,
                                d_hit_indices,
                                d_traced_rho);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        sort_rho_dists_tot += elapsed;

        /* End of full trace. */

        // Also profile the pure hit-count tracing.  Useful for optimizing the
        // pure tracing performance.
        hipEventRecord(part_start);
        grace::trace_hitcounts(d_rays,
                               d_ray_offsets,
                               d_tree,
                               d_spheres_xyzr);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&elapsed, part_start, part_stop);
        trace_hit_tot += elapsed;

        hipEventRecord(tot_stop);
        hipEventSynchronize(tot_stop);
        hipEventElapsedTime(&elapsed, tot_start, tot_stop);
        all_tot += elapsed;


        // Must be done in-loop for hipMemGetInfo to return relevant results.
        if (i == 0) {
            float trace_bytes = 0.0;
            float unused_bytes = 0.0;
            trace_bytes += d_rays.size() * sizeof(grace::Ray);
            trace_bytes += d_traced_rho.size() * sizeof(float);
            trace_bytes += d_ray_offsets.size() * sizeof(float);
            trace_bytes += d_hit_indices.size() * sizeof(unsigned int);
            trace_bytes += d_tree.nodes.size() * sizeof(int4);
            trace_bytes += d_tree.leaves.size() * sizeof(int4);
            trace_bytes += d_spheres_xyzr.size() * sizeof(float4);
            trace_bytes += d_rho.size() * sizeof(float);
            trace_bytes += grace::N_table * sizeof(float); // Integral lookup.
            trace_bytes += d_hit_distances.size() * sizeof(float);

            unused_bytes += d_keys.size() * sizeof(grace::uinteger32);
            unused_bytes += d_deltas.size() * sizeof(float);
            // Ray keys, used when generating rays.
            unused_bytes += d_rays.size() * sizeof(unsigned int);

            std::cout << "Total hits: " << d_hit_indices.size() << std::endl;
            std::cout << std::endl;
            std::cout << "Total memory for full trace kernel and sort: "
                      << trace_bytes / (1024.*1024.*1024.) << " GiB"
                      << std::endl;
            std::cout << "Allocated memory not used in trace kernel:   "
                      << unused_bytes / (1024.*1024.*1024.) << " GiB"
                      << std::endl;

            size_t avail, total;
            hipMemGetInfo(&avail, &total);
            std::cout << "Free memory:  " << avail / (1024.*1024.*1024.)
                      << " GiB" << std::endl;
            std::cout << "Total memory: " << total / (1024.*1024.*1024.)
                      << " GiB" << std::endl;
            std::cout << std::endl;
        }
    }

    std::cout << "Time for generating and sorting rays:   ";
    std::cout.width(8);
    std::cout << gen_ray_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for hit count tracing:             ";
    std::cout.width(8);
    std::cout << trace_hit_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for cumulative density tracing:    ";
    std::cout.width(8);
    std::cout << trace_rho_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for full tracing:                  ";
    std::cout.width(8);
    std::cout << trace_full_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for sort-by-distance:              ";
    std::cout.width(8);
    std::cout << sort_rho_dists_tot / N_iter << " ms" << std::endl;

    std::cout << "Time for total (inc. memory ops):       ";
    std::cout.width(8);
    std::cout << all_tot / N_iter << " ms" << std::endl;

    std::cout << std::endl;

} // End device code.

    // Exit cleanly to ensure full profiler trace.
    hipDeviceReset();
    return 0;
}
