#include "hip/hip_runtime.h"
#include <cmath>
#include <sstream>
#include <fstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "utils.cuh"
#include "../types.h"
#include "../nodes.h"
#include "../ray.h"
#include "../kernels/morton.cuh"
#include "../kernels/bintree_build.cuh"
#include "../kernels/bintree_trace.cuh"

int main(int argc, char* argv[])
{
    unsigned int N_rays = 250000;
    if (argc > 1) {
        N_rays = (unsigned int) std::strtol(argv[1], NULL, 10);
    }

    std::ifstream file;
    std::string fname = "Data_025";
    std::cout << "Reading in data from Gadget file " << fname << "..."
              << std::endl;

    // Read in gas data from Gadget-2 file.
    // Arrays are resized in read_gadget_gas().
    thrust::host_vector<float4> h_spheres_xyzr(1);
    thrust::host_vector<float> h_masses(1);
    thrust::host_vector<float> h_rho(1);

    file.open(fname.c_str(), std::ios::binary);
    grace::read_gadget_gas(file, h_spheres_xyzr, h_masses, h_rho);
    file.close();

    size_t N = h_spheres_xyzr.size();
    std::cout << "Will trace " << N_rays << " rays through " << N
              << " particles..." << std::endl;
    std::cout << std::endl;

    // Masses unused.
    h_masses.clear(); h_masses.shrink_to_fit();


// Device code.
{
    thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;
    thrust::device_vector<float> d_rho = h_rho;

    // Set the tree AABB.
    float min_x, max_x;
    grace::min_max_x(&min_x, &max_x, d_spheres_xyzr);

    float min_y, max_y;
    grace::min_max_y(&min_y, &max_y, d_spheres_xyzr);

    float min_z, max_z;
    grace::min_max_z(&min_z, &max_z, d_spheres_xyzr);

    float min_r, max_r;
    grace::min_max_w(&min_r, &max_r, d_spheres_xyzr);

    float4 bot = make_float4(min_x, min_y, min_z, 0.f);
    float4 top = make_float4(max_x, max_y, max_z, 0.f);

    // Generate morton keys based on particles' positions.
    thrust::device_vector<grace::uinteger32> d_keys(N);
    thrust::device_vector<grace::uinteger32> d_keys_2(N);
    grace::morton_keys(d_spheres_xyzr, d_keys, bot, top);
    thrust::copy(d_keys.begin(), d_keys.end(), d_keys_2.begin());

    // Sort particle positions and smoothing lengths by their keys.
    thrust::sort_by_key(d_keys_2.begin(), d_keys_2.end(),
                        d_spheres_xyzr.begin());
    d_keys_2.clear(); d_keys_2.shrink_to_fit();
    // Sort other properties by the same keys.
    thrust::device_vector<int> d_indices(N);
    thrust::device_vector<float> d_sorted(N);
    thrust::sequence(d_indices.begin(), d_indices.end());
    thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

    thrust::gather(d_indices.begin(), d_indices.end(),
                   d_rho.begin(), d_sorted.begin());
    d_rho = d_sorted;

    // Clear temporary storage.
    d_sorted.clear(); d_sorted.shrink_to_fit();
    d_indices.clear(); d_indices.shrink_to_fit();

    // Build the tree hierarchy from the keys.
    grace::Nodes d_nodes(N-1);
    grace::Leaves d_leaves(N);
    grace::build_nodes(d_nodes, d_leaves, d_keys);
    // Keys no longer needed.
    d_keys.clear(); d_keys.shrink_to_fit();
    grace::find_AABBs(d_nodes, d_leaves, d_spheres_xyzr);

        // Generate the rays (emitted from box centre (.5, .5, .5) of length 2).
    thrust::host_vector<grace::Ray> h_rays(N_rays);
    thrust::host_vector<float> h_dxs(N_rays);
    thrust::host_vector<float> h_dys(N_rays);
    thrust::host_vector<float> h_dzs(N_rays);
    thrust::host_vector<grace::uinteger32> h_keys(N_rays);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N_rays),
                      h_dxs.begin(),
                      grace::random_float_functor(0u, -1.0f, 1.0f) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N_rays),
                      h_dys.begin(),
                      grace::random_float_functor(1u, -1.0f, 1.0f) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N_rays),
                      h_dzs.begin(),
                      grace::random_float_functor(2u, -1.0f, 1.0f) );
    float x_centre = (max_x+min_x) / 2.;
    float y_centre = (max_y+min_y) / 2.;
    float z_centre = (max_z+min_z) / 2.;
    // Ensure rays end (well) outside box.
    float length = sqrt((max_x-min_x)*(max_x-min_x) +
                        (max_y-min_y)*(max_y-min_y) +
                        (max_z-min_z)*(max_z-min_z));
    for (int i=0; i<N_rays; i++) {
        float N_dir = sqrt(h_dxs[i]*h_dxs[i] +
                           h_dys[i]*h_dys[i] +
                           h_dzs[i]*h_dzs[i]);
        h_rays[i].dx = h_dxs[i] / N_dir;
        h_rays[i].dy = h_dys[i] / N_dir;
        h_rays[i].dz = h_dzs[i] / N_dir;
        h_rays[i].ox = x_centre;
        h_rays[i].oy = y_centre;
        h_rays[i].oz = z_centre;
        h_rays[i].length = length;
        h_rays[i].dclass = 0;
        if (h_dxs[i] >= 0)
            h_rays[i].dclass += 1;
        if (h_dys[i] >= 0)
            h_rays[i].dclass += 2;
        if (h_dzs[i] >= 0)
            h_rays[i].dclass += 4;

        // morton_key(float, float, float) requires floats in (0, 1).
        h_keys[i] = grace::morton_key((h_rays[i].dx+1)/2.f,
                                      (h_rays[i].dy+1)/2.f,
                                      (h_rays[i].dz+1)/2.f);
    }
    h_dxs.clear();
    h_dxs.shrink_to_fit();
    h_dys.clear();
    h_dys.shrink_to_fit();
    h_dzs.clear();
    h_dxs.shrink_to_fit();

    // Sort rays by Morton key and trace for per-ray hit couynts.
    thrust::sort_by_key(h_keys.begin(), h_keys.end(), h_rays.begin());
    thrust::device_vector<grace::Ray> d_rays = h_rays;
    thrust::device_vector<unsigned int> d_hit_counts(N_rays);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    grace::gpu::trace_hitcount<<<28, TRACE_THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(d_rays.data()),
        d_rays.size(),
        thrust::raw_pointer_cast(d_hit_counts.data()),
        thrust::raw_pointer_cast(d_nodes.left.data()),
        thrust::raw_pointer_cast(d_nodes.right.data()),
        thrust::raw_pointer_cast(d_nodes.AABB.data()),
        d_nodes.left.size(),
        thrust::raw_pointer_cast(d_spheres_xyzr.data()));
    CUDA_HANDLE_ERR( hipPeekAtLastError() );
    CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    int max_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                  0, thrust::maximum<unsigned int>());
    int min_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                  N+1, thrust::minimum<unsigned int>());
    float mean_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                     0, thrust::plus<unsigned int>())
                                    / float(N_rays);
    std::cout << "Time for hit-count tracing kernel: " << elapsed << " ms"
              << std::endl;


    thrust::device_vector<float> d_traced_rho(N_rays);
    // Copy tabulated kernel integrals to device.
    thrust::device_vector<float> d_b_integrals(grace::kernel_integral_table,
                                               grace::kernel_integral_table+51);
    // Trace and integrate through smoothing kernels, accumulating density.
    hipEventRecord(start);
    grace::gpu::trace_property<<<28, TRACE_THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(d_rays.data()),
        d_rays.size(),
        thrust::raw_pointer_cast(d_traced_rho.data()),
        thrust::raw_pointer_cast(d_nodes.left.data()),
        thrust::raw_pointer_cast(d_nodes.right.data()),
        thrust::raw_pointer_cast(d_nodes.AABB.data()),
        d_nodes.left.size(),
        thrust::raw_pointer_cast(d_spheres_xyzr.data()),
        thrust::raw_pointer_cast(d_rho.data()),
        thrust::raw_pointer_cast(d_b_integrals.data()));
    CUDA_HANDLE_ERR( hipPeekAtLastError() );
    CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    // Find min, max of output vector for 'plotting'.
    float max_rho = thrust::reduce(d_traced_rho.begin(), d_traced_rho.end(),
                                   0.0f, thrust::maximum<float>());
    float min_rho = thrust::reduce(d_traced_rho.begin(), d_traced_rho.end(),
                                   1E20, thrust::minimum<float>());
    float mean_rho = thrust::reduce(d_traced_rho.begin(), d_traced_rho.end(),
                                    0.0f, thrust::plus<float>()) / d_traced_rho.size();
    std::cout << "Time for acummulating integrating tracing kernel: "
              << elapsed << " ms" << std::endl;


    // Allocate output array based on per-ray hit counts, and calculate
    // individual ray offsets into this array.
    // int last_ray_hits = d_hit_counts[N_rays-1];
    // thrust::exclusive_scan(d_hit_counts.begin(), d_hit_counts.end(),
    //                        d_hit_counts.begin());
    // thrust::device_vector<float> d_trace_output(d_hit_counts[N_rays-1]+
    //                                             last_ray_hits);
    // thrust::device_vector<float> d_trace_distances(d_trace_output.size());

    // // Trace and integrate through smoothing kernels, accumulating density.
    // hipEventRecord(start);
    // grace::gpu::trace<<<28, TRACE_THREADS_PER_BLOCK>>>(
    //     thrust::raw_pointer_cast(d_rays.data()),
    //     d_rays.size(),
    //     thrust::raw_pointer_cast(d_trace_output.data()),
    //     thrust::raw_pointer_cast(d_trace_distances.data()),
    //     thrust::raw_pointer_cast(d_hit_counts.data()),
    //     thrust::raw_pointer_cast(d_nodes.data()),
    //     thrust::raw_pointer_cast(d_leaves.data()),
    //     d_nodes.size(),
    //     thrust::raw_pointer_cast(d_spheres_xyzr.data()),
    //     thrust::raw_pointer_cast(d_rho.data()),
    //     thrust::raw_pointer_cast(d_b_integrals.data()));
    // CUDA_HANDLE_ERR( hipPeekAtLastError() );
    // CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsed, start, stop);
    // std::cout << "Time for per-intersection integrating kernel: " << elapsed
    //           << " ms" << std::endl;

    // // Sort output arrays based on hit distances.
    // thrust::host_vector<int> h_hit_counts = d_hit_counts;
    // double t = 0.0;
    // for (int i=0; i<N_rays_side; i++) {
    //     int r_start = h_hit_counts[i];
    //     int r_end;
    //     if (i == N_rays-1)
    //         r_end = h_hit_counts[i] + last_ray_hits - 1;
    //     else
    //         r_end = h_hit_counts[i+1] - 1;
    //     hipEventRecord(start);
    //     thrust::sort_by_key(d_trace_distances.begin()+r_start,
    //                         d_trace_distances.begin()+r_end,
    //                         d_trace_output.begin()+r_start);
    //     hipEventRecord(stop);
    //     hipEventSynchronize(stop);
    //     hipEventElapsedTime(&elapsed, start, stop);
    //     t += elapsed;
    // }
    // std::cout << "Time for per-intersection sorting loop: " << t << " ms"
    //           << std::endl;
    std::cout << std::endl;

    std::cout << "Number of rays:       " << N_rays << std::endl;
    std::cout << "Number of particles:  " << N << std::endl;
    std::cout << "Mean hits:            " << mean_hits << std::endl;
    std::cout << "Max hits:             " << max_hits << std::endl;
    std::cout << "Min hits:             " << min_hits << std::endl;
    std::cout << "Mean output           " << mean_rho << std::endl;
    std::cout << "Max output:           " << max_rho << std::endl;
    std::cout << "Min output:           " << min_rho << std::endl;
    std::cout << std::endl;

} // End device code.  Call all thrust destructors etc. before hipDeviceReset().

    // Exit cleanly to ensure full profiler trace.
    hipDeviceReset();
    return 0;
}
