#include "hip/hip_runtime.h"
#include "tris_render.cuh"
#include "tris_trace.cuh"

#include "helper/vector_math.cuh"

#include "grace/cuda/kernel_config.h"

#include <algorithm>
#include <cmath>

void setup_lights(
    const float3 bots, const float3 tops,
    thrust::device_vector<float3>& d_lights_pos)
{
    float3 centre = make_float3((bots.x + tops.x) / 2.,
                                (bots.y + tops.y) / 2.,
                                (bots.z + tops.z) / 2.);
    float max_span = max(tops.x - bots.x,
                         max(tops.y - bots.y, tops.z - bots.z));

    // Above
    d_lights_pos.push_back(
        make_float3(centre.x, tops.y + max_span, tops.z + max_span)
    );
    // Left
    // d_lights_pos.push_back(
    //     make_float3(bots.x - max_span, centre.y, tops.z + max_span)
    // );
}

void setup_camera(
    const float3 bots, const float3 tops, const float FOVy_degrees,
    const int resolution_x, const int resolution_y,
    float3* camera_position, float3* look_at, float3* view_up,
    float* FOVy_radians, float* ray_length)
{
    const float3 size = make_float3(tops.x - bots.x,
                                    tops.y - bots.y,
                                    tops.z - bots.z);
    const float3 center = make_float3((bots.x + tops.x) / 2.,
                                      (bots.y + tops.y) / 2.,
                                      (bots.z + tops.z) / 2.);

    *look_at = center;
    *view_up = make_float3(0.f, 1.f, 0.f);
    *ray_length = 100. * size.z;

    *FOVy_radians = FOVy_degrees * 3.141 / 180.;

    // Compute the z-position of the camera, given the fixed field-of-view, such
    // that the entire bounding box will always be visible.
    float FOVx_radians = 2. * std::atan2(std::tan(*FOVy_radians / 2.),
                                         (double)resolution_x / resolution_y);
    float L_x = 1.02 * size.x / FOVx_radians;
    float L_y = 1.02 * size.y / *FOVy_radians;
    float camera_z = look_at->z + std::max(L_x, L_y);

    *camera_position = make_float3(bots.x - 0.1 * size.x,
                                   tops.y + 0.3 * size.y,
                                   camera_z);
}

static __global__ void shade_triangles_kernel(
    const Triangle* triangles,
    const size_t N_tris,
    const float3* const lights_pos,
    const size_t N_lights,
    float* const shaded_tris)
{
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x;
         tid < N_tris;
         tid += gridDim.x)
    {
        Triangle tri = triangles[tid];
        float3 normal = normalize(cross_product(tri.e1, tri.e2));

        for (int l = 0; l < N_lights; ++l)
        {
            float3 light_pos = lights_pos[l];
            float3 L = normalize(light_pos - tri.v);

            // The true value would vary with the point at which the ray
            // intersects the triangle. However, provided that
            // |L| >> |tri.e1|, |tri.e2| (i.e. the light is far away from the
            // triangle) the below is approximately correct.
            float shading = max(0.0, dot_product(L, normal));

            shaded_tris[l * N_tris + tid] = shading;
        }
    }
}

void shade_triangles(
    const thrust::device_vector<Triangle>& d_tris,
    const thrust::device_vector<float3>& d_lights_pos,
    thrust::device_vector<float>& d_shaded_tris)
{
    d_shaded_tris.resize(d_tris.size() * d_lights_pos.size());

    const int NT = 128;
    const int blocks = min((int)((d_tris.size() + NT - 1) / NT), grace::MAX_BLOCKS);
    shade_triangles_kernel<<<NT, blocks>>>(
        thrust::raw_pointer_cast(d_tris.data()),
        d_tris.size(),
        thrust::raw_pointer_cast(d_lights_pos.data()),
        d_lights_pos.size(),
        thrust::raw_pointer_cast(d_shaded_tris.data())
    );

}

static __global__ void shade_pixels_kernel(
    const PrimaryRayResult* const primary_results,
    const size_t N_primary,
    const ShadowRayResult* const shadow_results,
    const int N_shadow_per_primary,
    const float* const shaded_tris,
    float* const pixels)
{
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x;
         tid < N_primary;
         tid += gridDim.x)
    {
        float brightness = 0.f;
        for (int i = 0; i < N_shadow_per_primary; ++i)
        {
            int tri_idx = shadow_results[i * N_shadow_per_primary + tid].idx;
            if (tri_idx == -1)
            {
                // Not blocked.
                brightness = brightness + 1.f;
            }
        }
        brightness = AMBIENT_BKG + brightness / N_shadow_per_primary;

        float colour = BKG_COLOUR; // Assume miss.
        int tri_idx = primary_results[tid].idx;
        if (tri_idx != -1)
        {
            colour = brightness * shaded_tris[tri_idx];
        }

        pixels[tid] = colour;
    }
}

static void shade_pixels(
    const thrust::device_vector<PrimaryRayResult>& d_primary_results,
    const thrust::device_vector<ShadowRayResult>& d_shadow_results,
    const thrust::device_vector<float>& d_shaded_tris,
    thrust::device_vector<float>& d_pixels)
{
    const int NT = 128;
    const int blocks = min((int)((d_primary_results.size() + NT - 1) / NT), grace::MAX_BLOCKS);
    shade_pixels_kernel<<<blocks, NT>>>(
        thrust::raw_pointer_cast(d_primary_results.data()),
        d_primary_results.size(),
        thrust::raw_pointer_cast(d_shadow_results.data()),
        d_shadow_results.size() / d_primary_results.size(),
        thrust::raw_pointer_cast(d_shaded_tris.data()),
        thrust::raw_pointer_cast(d_pixels.data())
    );
}

void render(
    const thrust::device_vector<grace::Ray>& d_rays,
    const thrust::device_vector<Triangle>& d_tris,
    const grace::Tree& d_tree,
    const thrust::device_vector<float3>& d_lights_pos,
    const thrust::device_vector<float>& d_shaded_tris,
    thrust::device_vector<float>& d_pixels)
{
    thrust::device_vector<grace::Ray> d_shadow_rays(d_rays.size());
    thrust::device_vector<PrimaryRayResult> d_primary_results(d_rays.size());
    thrust::device_vector<ShadowRayResult>
        d_shadow_results(d_rays.size() * d_lights_pos.size());

    trace_primary_rays(d_rays, d_tris, d_tree, d_primary_results);

    // Trace shadow rays to each light source.
    for (int i = 0; i < d_lights_pos.size(); ++i)
    {
        generate_shadow_rays(i, d_lights_pos, d_rays, d_primary_results,
                             d_shadow_rays);

        trace_shadow_rays(d_shadow_rays, d_tris, d_tree,
                          d_shadow_results.data() + i * d_rays.size());
    }

    shade_pixels(d_primary_results, d_shadow_results, d_shaded_tris, d_pixels);
}
