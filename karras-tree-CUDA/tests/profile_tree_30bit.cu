#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "utils.cuh"
#include "../types.h"
#include "../nodes.h"
#include "../kernels/morton.cuh"
#include "../kernels/bintree_build_kernels.cuh"

int main(int argc, char* argv[]) {

    typedef grace::Vector3<float> Vector3f;
    hipDeviceProp_t deviceProp;
    std::ofstream outfile;
    std::string file_name;
    std::ostringstream converter;

    outfile.setf(std::ios::fixed, std::ios::floatfield);
    outfile.precision(5);
    outfile.fill('0');


    /* Initialize run parameters. */

    unsigned int levels = 20;
    unsigned int N_iter = 1000;
    unsigned int file_num = 1;
    unsigned int device_ID = 0;
    unsigned int seed_factor = 1u;
    if (argc > 5) {
        seed_factor = (unsigned int) std::strtol(argv[5], NULL, 10);
    }
    if (argc > 4) {
        levels = (unsigned int) std::strtol(argv[4], NULL, 10);
        // Keep levels in [5, 25].
        levels = min(25, max(5, levels));
    }
    if (argc > 3) {
        N_iter = (unsigned int) std::strtol(argv[3], NULL, 10);
    }
    if (argc > 2) {
        device_ID = (unsigned int) std::strtol(argv[2], NULL, 10);
    }
    if (argc > 1) {
        file_num = (unsigned int) std::strtol(argv[1], NULL, 10);

    }
    // Convert file number to a string.
    converter << file_num;
    file_name = ("profile_tree_" + converter.str() + ".log");

    unsigned int N = 1u << (levels - 1);
    std::cout << "Will profile (on device " << device_ID << " with " << N_iter
              << " iterations):" << std::endl;
    std::cout << "    i)  A tree constructed from " << N
              << " uniform random positions." << std::endl;
    std::cout << "    ii) AABB finding (only) of a fully balanced tree with "
              << N << " leaves." << std::endl;
    std::cout << std::endl;
    std::cout << "Saving results to " << file_name << std::endl;


    /* Write run parameters to file. */

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);
    // Wipe the file, if it exists.
    outfile.open(file_name.c_str(), std::ofstream::out | std::ofstream::trunc);
    outfile << "Device " << device_ID
                    << ":                 " << deviceProp.name << std::endl;
    outfile << "Tree depth:               " << levels << std::endl;
    outfile << "Number of leaves:         " << N << std::endl;
    outfile << "Number of nodes + leaves: " << 2*N - 1 << std::endl;
    outfile << "Iterations per tree:      " << N_iter << std::endl;
    outfile << "Random points' seed factor: " << seed_factor << std::endl;
    outfile << "MORTON_THREADS_PER_BLOCK: " << MORTON_THREADS_PER_BLOCK
            << std::endl;
    outfile << "BUILD_THREADS_PER_BLOCK:  " << BUILD_THREADS_PER_BLOCK
            << std::endl;
    outfile << "AABB_THREADS_PER_BLOCK:   " << AABB_THREADS_PER_BLOCK
            << std::endl;
    outfile << "MAX_BLOCKS:               " << MAX_BLOCKS << std::endl;
    outfile << std::endl << std::endl;
    outfile.close();


    /* Allocate arrays and generate input data. */

    // Generate N random positions and radii, i.e. 4N random floats in [0,1).
    thrust::host_vector<float> h_x_centres(N);
    thrust::host_vector<float> h_y_centres(N);
    thrust::host_vector<float> h_z_centres(N);
    thrust::host_vector<float> h_radii(N);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_x_centres.begin(),
                      random_float_functor(0u, seed_factor) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_y_centres.begin(),
                      random_float_functor(1u, seed_factor) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_z_centres.begin(),
                      random_float_functor(2u, seed_factor) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_radii.begin(),
                      random_float_functor(0.1f, seed_factor) );

    // Set the AABBs.
    Vector3f bottom(0., 0., 0.);
    Vector3f top(1., 1., 1.);


    /* Profile the tree constructed from random data. */

    hipEvent_t part_start, part_stop;
    hipEvent_t tot_start, tot_stop;
    float part_elapsed, tot_elapsed;
    float times[5];
    hipEventCreate(&part_start);
    hipEventCreate(&part_stop);
    hipEventCreate(&tot_start);
    hipEventCreate(&tot_stop);

    for (int i=0; i<N_iter; i++) {
        hipEventRecord(tot_start);
        // Copy pristine host-side data to GPU.
        thrust::device_vector<float> d_x_centres = h_x_centres;
        thrust::device_vector<float> d_y_centres = h_y_centres;
        thrust::device_vector<float> d_z_centres = h_z_centres;
        thrust::device_vector<float> d_radii = h_radii;

        // Generate the Morton keys for each position.
        thrust::device_vector<UInteger32> d_keys(N);
        hipEventRecord(part_start);
        grace::morton_keys(d_x_centres, d_y_centres, d_z_centres,
                           d_keys, bottom, top);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        times[1] += part_elapsed;

        // Sort the positions by their keys and save the sorted keys.
        thrust::device_vector<int> d_indices(N);
        thrust::device_vector<float> d_tmp(N);
        hipEventRecord(part_start);
        thrust::sequence(d_indices.begin(), d_indices.end());
        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_x_centres.begin(),
                       d_tmp.begin());
        d_x_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_y_centres.begin(),
                       d_tmp.begin());
        d_y_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_z_centres.begin(),
                       d_tmp.begin());
        d_z_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_radii.begin(),
                       d_tmp.begin());
        d_radii = d_tmp;
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        times[2] += part_elapsed;


        // Build the tree hierarchy from the keys.
        thrust::device_vector<grace::Node> d_nodes(N-1);
        thrust::device_vector<grace::Leaf> d_leaves(N);
        hipEventRecord(part_start);
        grace::build_nodes(d_nodes, d_leaves, d_keys);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        times[3] += part_elapsed;

        // Find the AABBs.
        hipEventRecord(part_start);
        grace::find_AABBs(d_nodes, d_leaves,
                          d_x_centres, d_y_centres, d_z_centres, d_radii);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        times[4] += part_elapsed;

        // Record the total time spent in the loop.
        hipEventRecord(tot_stop);
        hipEventSynchronize(tot_stop);
        hipEventElapsedTime(&tot_elapsed, tot_start, tot_stop);
        times[0] += tot_elapsed;
    }


    /* Calculate mean times and write results to file. */

    for (int i=0; i<5; i++) {
        times[i] /= N_iter;
    }

    outfile.open(file_name.c_str(), std::ofstream::out | std::ofstream::app);
    outfile << "Time for morton key generation: " << times[1] << " ms."
            << std::endl;
    outfile << "Time for sort-by-key:           " << times[2] << " ms."
            << std::endl;
    outfile << "Time for hierarchy generation:  " << times[3] << " ms."
            << std::endl;
    outfile << "Time for calculating AABBs:     " << times[4] << " ms."
            << std::endl;
    outfile << "Total time for loop:            " << times[0] << " ms."
            << std::endl;
    outfile << std::endl << std::endl;
    outfile.close();
}
