#include "hip/hip_runtime.h"
#include "williams.cuh"

__host__ __device__ int williams(const Ray& ray, const AABB& box)
{
    float bx, by, bz;
    float tx, ty, tz;

    if (ray.invdx >= 0) {
        bx = (box.bx - ray.ox) * ray.invdx;
        tx = (box.tx - ray.ox) * ray.invdx;
    }
    else {
        bx = (box.tx - ray.ox) * ray.invdx;
        tx = (box.bx - ray.ox) * ray.invdx;
    }
    if (ray.invdy >= 0) {
        by = (box.by - ray.oy) * ray.invdy;
        ty = (box.ty - ray.oy) * ray.invdy;
    }
    else {
        by = (box.ty - ray.oy) * ray.invdy;
        ty = (box.by - ray.oy) * ray.invdy;
    }
    if (ray.invdz >= 0) {
        bz = (box.bz - ray.oz) * ray.invdz;
        tz = (box.tz - ray.oz) * ray.invdz;
    }
    else {
        bz = (box.tz - ray.oz) * ray.invdz;
        tz = (box.bz - ray.oz) * ray.invdz;
    }

    float tmin, tmax;
    tmin = fmax( fmax(bx, by), fmax(bz, 0) );
    tmax = fmin( fmin(tx, ty), fmin(tz, ray.length) );

    return (tmax >= tmin ? HIT : MISS);
}

__host__ __device__ int williams_noif(const Ray& ray, const AABB& box)
{
    float bx = box.bx;
    float by = box.by;
    float bz = box.bz;
    float tx = box.tx;
    float ty = box.ty;
    float tz = box.tz;

    bx = (bx - ray.ox) * ray.invdx;
    tx = (tx - ray.ox) * ray.invdx;
    by = (by - ray.oy) * ray.invdy;
    ty = (ty - ray.oy) * ray.invdy;
    bz = (bz - ray.oz) * ray.invdz;
    tz = (tz - ray.oz) * ray.invdz;

    float tmin, tmax;
    tmin = fmax( fmax(fmin(bx, tx), fmin(by, ty)), fmax(fmin(bz, tz), 0) );
    tmax = fmin( fmin(fmax(bx, tx), fmax(by, ty)), fmin(fmax(bz, tz), ray.length) );

    return (tmax >= tmin ? HIT : MISS);
}
