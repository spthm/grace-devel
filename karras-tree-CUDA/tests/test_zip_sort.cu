#include <iostream>
#include <cstdlib>

#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>

#include "../types.h"
#include "../kernels/morton.cuh"

__host__ __device__ unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

class random_pos_functor
{
    const int offset;

public:
    random_pos_functor(const int offset_) : offset(offset_) {}

    __host__ __device__ float operator() (unsigned int n)
    {
        unsigned int seed = hash(3*n);
        thrust::default_random_engine rng(seed);
        thrust::uniform_real_distribution<float> u01(0,1);

        for (int i=0; i<offset; i++) {
            rng.discard(1);
        }

        return u01(rng);
    }
};


int main(int argc, char* argv[]) {

    typedef grace::Vector3<float> Vector3f;
    hipEvent_t start, stop;
    float elapsed_time;
    float total_time_vector3 = 0;
    float total_time_zip = 0;
    float total_time_gather = 0;


    /* Generate N random positions, i.e. 3*N random floats in [0,1) */

    unsigned int N;
    unsigned int Niter;
    if (argc > 1)
        N = (unsigned int) std::strtol(argv[1], NULL, 10);
    else
        N = 1000000;
    if (argc > 2)
        Niter = (unsigned int) std::strtol(argv[2], NULL, 10);
    else
        Niter = 10000;
    std::cout << "Will generate " << N << " random points for " << Niter
              << " iteration" << ((Niter > 1) ? "s" : "") << "...\n"
              << std::endl;

    thrust::host_vector<float> h_x_centres(N);
    thrust::host_vector<float> h_y_centres(N);
    thrust::host_vector<float> h_z_centres(N);

    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_x_centres.begin(),
                      random_pos_functor(0) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_y_centres.begin(),
                      random_pos_functor(1) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_z_centres.begin(),
                      random_pos_functor(2) );


    /* Copy centres into a host vector of Vector3s. */

    thrust::host_vector<Vector3f> h_centres(N);
    for (int i=0; i<N; i++) {
        h_centres[i].x = h_x_centres[i];
        h_centres[i].y = h_y_centres[i];
        h_centres[i].z = h_z_centres[i];
    }


    /* Generate the Morton key of each position. */

    thrust::device_vector<float> d_x_centres = h_x_centres;
    thrust::device_vector<float> d_y_centres = h_y_centres;
    thrust::device_vector<float> d_z_centres = h_z_centres;
    thrust::device_vector<UInteger32> d_keys(N);
    Vector3f bottom(0., 0., 0.);
    Vector3f top(1., 1., 1.);

    grace::morton_keys(d_x_centres, d_y_centres, d_z_centres,
                       d_keys, bottom, top);

    thrust::host_vector<float> h_keys = d_keys;


    /* Sort the centres vector, and the separate x, y, z vectors using a zip
     * iterator and an indices + gather method.  Repeat and record total time
     * taken for each method.
     */


    /* Measure time for sorting the Vector3. */

    thrust::device_vector<Vector3f> d_centres = h_centres;
    std::cout << "Running Vector3 sort iterations..." << std::endl;
    for (int i=0; i<Niter; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_centres.begin());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time_vector3 += elapsed_time;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        d_centres = h_centres;
        d_keys = h_keys;
    }


    /* Measure time for sorting the  zip iterator. */

    std::cout << "Running zip iterator sort iterations..." << std::endl;
    for (int i=0; i<Niter; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        thrust::sort_by_key(d_keys.begin(),
                            d_keys.end(),
                            thrust::make_zip_iterator(
                                thrust::make_tuple(d_x_centres.begin(),
                                                   d_y_centres.begin(),
                                                   d_z_centres.begin() ))
                            );

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time_zip += elapsed_time;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        d_x_centres = h_x_centres;
        d_y_centres = h_y_centres;
        d_z_centres = h_z_centres;
        d_keys = h_keys;
    }


    /* Measure time for sorting the gather method. */

    std::cout << "Running index sort and gather iterations..." << std::endl;
    thrust::device_vector<int> d_indices(N);
    thrust::device_vector<float> d_tmp(N);
    for (int i=0; i<Niter; i++) {
        thrust::sequence(d_indices.begin(), d_indices.end());

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_x_centres.begin(),
                       d_tmp.begin());
        d_x_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_y_centres.begin(),
                       d_tmp.begin());
        d_y_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_z_centres.begin(),
                       d_tmp.begin());
        d_z_centres = d_tmp;

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time_gather += elapsed_time;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        d_x_centres = h_x_centres;
        d_y_centres = h_y_centres;
        d_z_centres = h_z_centres;
        d_keys = h_keys;
    }

    std::cout << "Mean time taken for Vector3:            "
              << total_time_vector3 / (float) Niter << " ms." << std::endl;
    std::cout << "Mean time taken for zip iterator:       "
              << total_time_zip / (float) Niter << " ms." << std::endl;
    std::cout << "Mean time taken for indices and gather: "
              << total_time_gather / (float) Niter << " ms." << std::endl;

}
