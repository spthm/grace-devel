#include "hip/hip_runtime.h"
#include <cmath>
#include <sstream>
#include <fstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "utils.cuh"
#include "../types.h"
#include "../nodes.h"
#include "../ray.h"
#include "../kernels/morton.cuh"
#include "../kernels/bintree_build_kernels.cuh"
#include "../kernels/bintree_trace.cuh"

int main(int argc, char* argv[])
{
    typedef grace::Vector3<float> Vector3f;

    unsigned int N_rays = 250000;
    if (argc > 1) {
        N_rays = (unsigned int) std::strtol(argv[1], NULL, 10);
    }
    float N_rays_side = floor(pow(N_rays, 0.500001));

    std::ifstream file;
    std::string fname = "Data_025";
    std::cout << "Reading in data from Gadget file " << fname << "..."
              << std::endl;

    // Read in gas data from Gadget-2 file.
    // Arrays are resized in read_gadget_gas().
    thrust::host_vector<float> h_x_centres(1);
    thrust::host_vector<float> h_y_centres(1);
    thrust::host_vector<float> h_z_centres(1);
    thrust::host_vector<float> h_radii(1);
    thrust::host_vector<float> h_masses(1);

    file.open(fname.c_str(), std::ios::binary);
    read_gadget_gas(file, h_x_centres, h_y_centres, h_z_centres,
                          h_radii, h_masses);
    file.close();
    // Masses unused.
    h_masses.clear(); h_masses.shrink_to_fit();

    unsigned int N = h_x_centres.size();
    // Factor of two is a fudge.
    unsigned int N_hits_per_ray = floor(pow(N, 0.333334));
    std::cout << "Will trace " << N_rays << " rays through " << N
              << " particles..." << std::endl;
    std::cout << std::endl;

// Device code.
{
    thrust::device_vector<float> d_x_centres = h_x_centres;
    thrust::device_vector<float> d_y_centres = h_y_centres;
    thrust::device_vector<float> d_z_centres = h_z_centres;
    thrust::device_vector<float> d_radii = h_radii;

    // Set the AABBs.
    float max_x = thrust::reduce(h_x_centres.begin(),
                                 h_x_centres.end(),
                                 -1.0f,
                                 thrust::maximum<float>());
    float max_y = thrust::reduce(h_y_centres.begin(),
                                 h_y_centres.end(),
                                 -1.0f,
                                 thrust::maximum<float>());
    float max_z = thrust::reduce(h_z_centres.begin(),
                                 h_z_centres.end(),
                                 -1.0f,
                                 thrust::maximum<float>());
    float min_x = thrust::reduce(h_x_centres.begin(),
                                 h_x_centres.end(),
                                 max_x,
                                 thrust::minimum<float>());
    float min_y = thrust::reduce(h_y_centres.begin(),
                                 h_y_centres.end(),
                                 max_y,
                                 thrust::minimum<float>());
    float min_z = thrust::reduce(h_z_centres.begin(),
                                 h_z_centres.end(),
                                 max_z,
                                 thrust::minimum<float>());
    float max_r = thrust::reduce(h_radii.begin(),
                                 h_radii.end(),
                                 -1.0f,
                                 thrust::minimum<float>());
    Vector3f bottom(min_x, min_y, min_z);
    Vector3f top(max_x, max_y, max_z);

    // Generate morton keys based on particles' positions.
    thrust::device_vector<UInteger32> d_keys(N);
    grace::morton_keys(d_x_centres, d_y_centres, d_z_centres, d_keys,
                       bottom, top);

    // Sort all particle arrays by their keys.
    thrust::device_vector<int> d_indices(N);
    thrust::device_vector<float> d_tmp(N);
    thrust::sequence(d_indices.begin(), d_indices.end());
    thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

    thrust::gather(d_indices.begin(),
                   d_indices.end(),
                   d_x_centres.begin(),
                   d_tmp.begin());
    d_x_centres = d_tmp;

    thrust::gather(d_indices.begin(),
                   d_indices.end(),
                   d_y_centres.begin(),
                   d_tmp.begin());
    d_y_centres = d_tmp;

    thrust::gather(d_indices.begin(),
                   d_indices.end(),
                   d_z_centres.begin(),
                   d_tmp.begin());
    d_z_centres = d_tmp;

    thrust::gather(d_indices.begin(),
                   d_indices.end(),
                   d_radii.begin(),
                   d_tmp.begin());
    d_radii = d_tmp;
    // Clear temporary storage.
    d_tmp.clear(); d_tmp.shrink_to_fit();
    d_indices.clear(); d_indices.shrink_to_fit();

    // Build the tree hierarchy from the keys.
    thrust::device_vector<grace::Node> d_nodes(N-1);
    thrust::device_vector<grace::Leaf> d_leaves(N);
    grace::build_nodes(d_nodes, d_leaves, d_keys);
    // Keys no longer needed.
    d_keys.clear(); d_keys.shrink_to_fit();
    grace::find_AABBs(d_nodes, d_leaves,
                      d_x_centres, d_y_centres, d_z_centres, d_radii);

    // Generate the rays, emitted from box side (X, Y, min_z-max_r) and of
    // length (max_z + max_r) - (min_z - max_r).
    float span_x = 2*max_r + max_x - min_x;
    float span_y = 2*max_r + max_y - min_y;
    float span_z = 2*max_r + max_z - min_z;
    float spacer_x = span_x / N_rays_side;
    float spacer_y = span_y / N_rays_side;
    thrust::host_vector<grace::Ray> h_rays(N_rays);
    thrust::host_vector<UInteger32> h_keys(N_rays);
    int i, j;
    float ox, oy;
    for (i=0, ox=(min_x-max_r); i<N_rays_side; ox+=spacer_x, i++)
    {
        for (j=0, oy=(min_y-max_r); j<N_rays_side; oy+=spacer_y, j++)
        {
            // All rays point in +ve z direction.
            h_rays[i*N_rays_side +j].dx = 0.0f;
            h_rays[i*N_rays_side +j].dy = 0.0f;
            h_rays[i*N_rays_side +j].dz = 1.0f;

            h_rays[i*N_rays_side +j].ox = ox;
            h_rays[i*N_rays_side +j].oy = oy;
            h_rays[i*N_rays_side +j].oz = min_z - max_r;

            h_rays[i*N_rays_side +j].length = span_z;
            h_rays[i*N_rays_side +j].dclass = 7;
        }

        // Since all rays are PPP, base key on origin instead.
        // morton_key(float, float, float) requires floats in (0, 1).
        h_keys[i] = grace::morton_key((ox-(min_x-max_r))/span_x,
                                      (oy-(min_y-max_r))/span_y,
                                      0.0f);
    }

    // Sort rays by Morton key.
    thrust::sort_by_key(h_keys.begin(), h_keys.end(), h_rays.begin());
    thrust::device_vector<grace::Ray> d_rays = h_rays;
    thrust::device_vector<int> d_hits(N_hits_per_ray*N_rays);
    thrust::device_vector<int> d_hit_count(N_rays);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    grace::gpu::trace<<<28, TRACE_THREADS_PER_BLOCK>>>
                     (thrust::raw_pointer_cast(d_rays.data()),
                      d_rays.size(),
                      N_hits_per_ray,
                      thrust::raw_pointer_cast(d_hits.data()),
                      thrust::raw_pointer_cast(d_hit_count.data()),
                      thrust::raw_pointer_cast(d_nodes.data()),
                      thrust::raw_pointer_cast(d_leaves.data()),
                      thrust::raw_pointer_cast(d_x_centres.data()),
                      thrust::raw_pointer_cast(d_y_centres.data()),
                      thrust::raw_pointer_cast(d_z_centres.data()),
                      thrust::raw_pointer_cast(d_radii.data()));
    CUDA_HANDLE_ERR( hipPeekAtLastError() );
    CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    int max_hits = thrust::reduce(d_hit_count.begin(), d_hit_count.end(),
                                  0, thrust::maximum<int>());
    int min_hits = thrust::reduce(d_hit_count.begin(), d_hit_count.end(),
                                  N, thrust::minimum<int>());
    float mean_hits = thrust::reduce(d_hit_count.begin(), d_hit_count.end(),
                                     0, thrust::plus<int>()) / float(N_rays);
    std::cout << "Time for tracing kernel: " << elapsed << " ms" << std::endl;
    std::cout << std::endl;
    std::cout << "Number of rays:       " << N_rays << std::endl;
    std::cout << "Number of particles:  " << N << std::endl;
    std::cout << "Expected hit count:   " << N_hits_per_ray << std::endl;
    std::cout << "Mean hits:            " << mean_hits << std::endl;
    std::cout << "Max hits:             " << max_hits << std::endl;
    std::cout << "Min hits:             " << min_hits << std::endl;

    // Sort ray hit data such that increasing the index moves us along x first,
    // then y.
    thrust::host_vector<float> h_pos_keys(N_rays);
    for (int i=0; i<N_rays; i++) {
        h_pos_keys[i] = h_rays[i].ox + (2*span_x)*h_rays[i].oy;
    }
    thrust::host_vector<float> h_hit_count = d_hit_count;
    thrust::sort_by_key(h_pos_keys.begin(), h_pos_keys.end(),
                        h_hit_count.begin());
    for (int i=0; i<N_rays; i++) {
        h_pos_keys[i] = h_rays[i].ox + (2*span_x)*h_rays[i].oy;
    }
    thrust::sort_by_key(h_pos_keys.begin(), h_pos_keys.end(),
                        h_rays.begin());

    FILE *f;
    unsigned char *img = NULL;
    int w=N_rays_side;
    int h=N_rays_side;
    int filesize = 54 + 3*w*h;  //w is your image width, h is image height, both int
    if( img )
        free( img );
    img = (unsigned char *)malloc(3*w*h);
    memset(img,0,sizeof(img));

    int r, g, b, x, y;
    for(int i=0; i<w; i++)
    {
        for(int j=0; j<h; j++)
    {
        x=i; y=(h-1)-j;
        r = g = b = (int) (h_hit_count[i+w*j]*255.0/max_hits);
        if (r > 255) r=255;
        if (g > 255) g=255;
        if (b > 255) b=255;
        img[(x+y*w)*3+2] = (unsigned char)(r);
        img[(x+y*w)*3+1] = (unsigned char)(g);
        img[(x+y*w)*3+0] = (unsigned char)(b);
        //std::cout << "ox, oy: " << h_rays[i+w*j].ox << ", " << h_rays[i+w*j].oy
        //          << std::endl;
    }
    }

    unsigned char bmpfileheader[14] = {'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0};
    unsigned char bmpinfoheader[40] = {40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0};
    unsigned char bmppad[3] = {0,0,0};

    bmpfileheader[ 2] = (unsigned char)(filesize    );
    bmpfileheader[ 3] = (unsigned char)(filesize>> 8);
    bmpfileheader[ 4] = (unsigned char)(filesize>>16);
    bmpfileheader[ 5] = (unsigned char)(filesize>>24);

    bmpinfoheader[ 4] = (unsigned char)(       w    );
    bmpinfoheader[ 5] = (unsigned char)(       w>> 8);
    bmpinfoheader[ 6] = (unsigned char)(       w>>16);
    bmpinfoheader[ 7] = (unsigned char)(       w>>24);
    bmpinfoheader[ 8] = (unsigned char)(       h    );
    bmpinfoheader[ 9] = (unsigned char)(       h>> 8);
    bmpinfoheader[10] = (unsigned char)(       h>>16);
    bmpinfoheader[11] = (unsigned char)(       h>>24);

    f = fopen("img.bmp","wb");
    fwrite(bmpfileheader,1,14,f);
    fwrite(bmpinfoheader,1,40,f);
    for(i=0; i<h; i++)
    {
        fwrite(img+(w*(h-i-1)*3),3,w,f);
        fwrite(bmppad,1,(4-(w*3)%4)%4,f);
    }
    fclose(f);
} // Device code.  Call all thrust destructors etc. before hipDeviceReset().

    // Exit cleanly to ensure full profiler trace.
    hipDeviceReset();
    return 0;
}
