#include <iostream>
#include <cstdlib>

#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>

#include "../types.h"
#include "../kernels/morton.cuh"

__host__ __device__ unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

class random_vector3_functor
{
    // Constructed on the host.
    grace::Vector3<float> random_vector3;

public:
    __host__ __device__ grace::Vector3<float> operator() (unsigned int n)
    {
        unsigned int seed = hash(n);
        thrust::default_random_engine rng(seed);
        thrust::uniform_real_distribution<float> u01(0,1);


        random_vector3.x = u01(rng);
        random_vector3.y = u01(rng);
        random_vector3.z = u01(rng);

        return random_vector3;
    }
};


int main(int argc, char* argv[]) {

    typedef grace::Vector3<float> Vector3f;
    hipEvent_t start, stop;
    float elapsed_time;
    float total_time_vector3 = 0;
    float total_time_zip = 0;
    float total_time_gather = 0;


    /* Generate N random position vectors, i.e. 3*N random floats in [0,1) */

    unsigned int N;
    unsigned int Niter;
    if (argc > 1)
        N = (unsigned int) std::strtol(argv[1], NULL, 10);
    else
        N = 1000000;
    if (argc > 2)
        Niter = (unsigned int) std::strtol(argv[2], NULL, 10);
    else
        Niter = 10000;
    std::cout << "Will generate " << N << " random points for " << Niter
              << " iteration" << ((Niter > 1) ? "s" : "") << "...\n" << std::endl;

    thrust::host_vector<Vector3f> h_centres(N);

    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      h_centres.begin(),
                      random_vector3_functor() );


    /* Copy centres vector into separate x, y, z vectors. */

    thrust::host_vector<float> h_x_centres(N);
    thrust::host_vector<float> h_y_centres(N);
    thrust::host_vector<float> h_z_centres(N);

    for (int i=0; i<N; i++) {
        h_x_centres[i] = h_centres[i].x;
        h_y_centres[i] = h_centres[i].y;
        h_z_centres[i] = h_centres[i].z;
    }


    /* Generate the Morton key of each position. */

    thrust::host_vector<UInteger32> h_keys(N);
    Vector3f bottom(0., 0., 0.);
    Vector3f top(1., 1., 1.);

    thrust::transform(h_centres.begin(),
                      h_centres.end(),
                      h_keys.begin(),
                      grace::morton_key_functor<UInteger32, float>(bottom, top));


    /* Sort the centres vector, and the separate x, y, z vectors using a zip
     * iterator and an indices + gather method.  Repeat and record total time
     * taken for each method.
     */

    thrust::device_vector<Vector3f> d_centres;
    thrust::device_vector<float> d_x_centres;
    thrust::device_vector<float> d_y_centres;
    thrust::device_vector<float> d_z_centres;
    thrust::device_vector<float> d_keys;


    /* Measure time for sorting the Vector3. */

    std::cout << "Running Vector3 sort iterations..." << std::endl;
    for (int i=0; i<Niter; i++) {
        d_centres = h_centres;
        d_keys = h_keys;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_centres.begin());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time_vector3 += elapsed_time;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }


    /* Measure time for sorting the  zip iterator. */

    std::cout << "Running zip iterator sort iterations..." << std::endl;
    for (int i=0; i<Niter; i++) {
        d_x_centres = h_x_centres;
        d_y_centres = h_y_centres;
        d_z_centres = h_z_centres;
        d_keys = h_keys;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        thrust::sort_by_key(d_keys.begin(),
                            d_keys.end(),
                            thrust::make_zip_iterator(
                                thrust::make_tuple(d_x_centres.begin(),
                                                   d_y_centres.begin(),
                                                   d_z_centres.begin() ))
                            );

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time_zip += elapsed_time;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }


    /* Measure time for sorting the gather method. */

    std::cout << "Running index sort and gather iterations..." << std::endl;
    thrust::device_vector<int> d_indices(N);
    thrust::device_vector<float> d_tmp(N);
    for (int i=0; i<Niter; i++) {
        d_x_centres = h_x_centres;
        d_y_centres = h_y_centres;
        d_z_centres = h_z_centres;
        d_keys = h_keys;

        thrust::sequence(d_indices.begin(), d_indices.end());

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_indices.begin());

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_x_centres.begin(),
                       d_tmp.begin());
        d_x_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_y_centres.begin(),
                       d_tmp.begin());
        d_y_centres = d_tmp;

        thrust::gather(d_indices.begin(),
                       d_indices.end(),
                       d_z_centres.begin(),
                       d_tmp.begin());
        d_z_centres = d_tmp;

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsed_time, start, stop);
        total_time_gather += elapsed_time;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    std::cout << "Mean time taken for Vector3:            "
              << total_time_vector3 / (float) Niter << " ms." << std::endl;
    std::cout << "Mean time taken for zip iterator:       "
              << total_time_zip / (float) Niter << " ms." << std::endl;
    std::cout << "Mean time taken for indices and gather: "
              << total_time_gather / (float) Niter << " ms." << std::endl;

}
