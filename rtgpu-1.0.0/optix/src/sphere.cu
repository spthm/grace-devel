#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>

rtDeclareVariable(float3, center,,);
rtDeclareVariable(float,  radius,,);

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template<bool use_robust_method> 
__device__ void intersect_sphere(void)
{
    float3 O = ray.origin - center;
    float3 D = ray.direction;

    float b = dot(O, D);
    float c = dot(O, O)-radius*radius;
    float disc = b*b-c;
    if(disc > 0.0f)
    {
	float sdisc = sqrtf(disc);
	float root1 = (-b - sdisc);

	bool do_refine = false;

	float root11 = 0.0f;

	if(use_robust_method && fabsf(root1) > 10.f * radius) 
	    do_refine = true;

	if(do_refine) 
	{
	    // refine root1
	    float3 O1 = O + root1 * ray.direction;
	    b = dot(O1, D);
	    c = dot(O1, O1) - radius*radius;
	    disc = b*b - c;

	    if(disc > 0.0f) 
	    {
		sdisc = sqrtf(disc);
		root11 = (-b - sdisc);
	    }
	}

	bool check_second = true;
	if( rtPotentialIntersection( root1 + root11 ) ) 
	{
	    shading_normal = geometric_normal = (O + (root1 + root11)*D)/radius;
	    if(rtReportIntersection(0))
		check_second = false;
	} 

	if(check_second) 
	{
	    float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
	    if( rtPotentialIntersection( root2 ) ) 
	    {
		shading_normal = geometric_normal = (O + root2*D)/radius;
		rtReportIntersection(0);
	    }
	}
    }
}


RT_PROGRAM void intersect(int primIdx)
{
    intersect_sphere<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
    intersect_sphere<true>();
}

RT_PROGRAM void bounds(int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = center - radius;
    aabb->m_max = center + radius;
}
