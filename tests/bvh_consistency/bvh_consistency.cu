#include "hip/hip_runtime.h"
// Due to a bug in thrust, this must appear before thrust/sort.h
// The simplest solution is to put it here, despite already being included in
// all of the includes which require it.
// See http://stackoverflow.com/questions/23352122
#include <hiprand/hiprand_kernel.h>

#include "grace/cuda/nodes.h"
#include "grace/aabb.h"
#include "grace/sphere.h"
#include "helper/random.cuh"
#include "helper/tree.cuh"

#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cstdlib>
#include <iomanip>
#include <iostream>

typedef grace::Sphere<float> SphereType;

int main(int argc, char* argv[])
{
    hipDeviceProp_t deviceProp;
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(3);

    int max_per_leaf = 32;
    size_t N = 128 * 128 * 128;
    unsigned int device_ID = 0;

    if (argc > 1) {
        max_per_leaf = (int)std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        N = (size_t)std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        device_ID = (unsigned int)std::strtol(argv[3], NULL, 10);
    }

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);

    std::cout << "Max particles per leaf:   " << max_per_leaf << std::endl
              << "Number of particles:      " << N << std::endl
              << "Running on device:        " << device_ID
                                            << " (" << deviceProp.name << ")"
                                            << std::endl
              << std::endl;

    SphereType high = SphereType(1.0f, 1.0f, 1.0f, 0.1f);
    SphereType low = SphereType(-1.0f, -1.0f, -1.0f, 0.0f);

    thrust::device_vector<SphereType> d_spheres(N);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      d_spheres.begin(),
                      random_sphere_functor<SphereType>(low, high));
    grace::Tree d_tree(N, max_per_leaf);
    build_tree(d_spheres, d_tree);

    grace::H_Tree h_tree = d_tree;
    const int N_leaves = h_tree.size();
    const int N_nodes = N_leaves - 1;

    thrust::host_vector<int> parent_flags(N_nodes);
    thrust::host_vector<int> child_flags(N_nodes + N_leaves);
    thrust::host_vector<int> particle_flags(N);

    for (size_t ni = 0; ni < N_nodes; ++ni)
    {
        int4 node = h_tree.nodes[4 * ni];
        int l = node.x;
        int r = node.y;

        parent_flags[ni] += 1;
        child_flags[l] += 1;
        child_flags[r] += 1;
    }

    for (size_t li = 0; li < N_leaves; ++li)
    {
        int4 leaf = h_tree.leaves[li];
        int first = leaf.x;
        int size = leaf.y;

        for (int pi = first; pi < first + size; ++pi)
        {
            particle_flags[pi] += 1;
        }
    }

    size_t failures = 0;

    for (size_t ni = 0; ni < parent_flags.size(); ++ni)
    {
        int flag = parent_flags[ni];
        if (flag != 1) {
            std::cout << "Error: parent count @ " << ni << " = " << flag
                      << std::endl;
            failures += 1;
        }
    }

    for (size_t ci = 0; ci < child_flags.size(); ++ci)
    {
        int flag = child_flags[ci];

        if (ci == h_tree.root_index && flag != 0)
        {
            std::cout << "Error: child count @ root " << ci << " = " << flag
                      << std::endl;
            failures += 1;
        }

        if (ci != h_tree.root_index && flag != 1) {
            std::cout << "Error: child count @ " << ci << " = " << flag
                      << std::endl;
            failures += 1;
        }
    }

    for (size_t pi = 0; pi < N; ++pi)
    {
        int flag = particle_flags[pi];
        if (flag != 1) {
            std::cout << "Error: particle count @ " << pi << " = " << flag
                      << std::endl;
            failures += 1;
        }
    }

    if (failures == 0)
    {
        std::cout << "PASSED" << std::endl;
        return EXIT_SUCCESS;
    }
    else
    {
        std::cout << "FAILED" << std::endl;
        return EXIT_FAILURE;
    }
}
