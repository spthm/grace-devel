// Due to a bug in thrust, this must appear before thrust/sort.h
// The simplest solution is to put it here, despite already being included in
// all of the includes which require it.
// See http://stackoverflow.com/questions/23352122
#include <hiprand/hiprand_kernel.h>

#include "grace/cpp/bvh.h"
#include "grace/cuda/bvh.cuh"
#include "grace/aabb.h"
#include "grace/sphere.h"
#include "helper/random.cuh"
#include "helper/tree.cuh"

#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cstdlib>
#include <iomanip>
#include <iostream>

typedef grace::Sphere<float> SphereType;

int main(int argc, char* argv[])
{
    hipDeviceProp_t deviceProp;
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(3);

    int max_per_leaf = 32;
    size_t N = 128 * 128 * 128;
    unsigned int device_ID = 0;

    if (argc > 1) {
        max_per_leaf = (int)std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        N = (size_t)std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        device_ID = (unsigned int)std::strtol(argv[3], NULL, 10);
    }

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);

    std::cout << "Max particles per leaf:   " << max_per_leaf << std::endl
              << "Number of particles:      " << N << std::endl
              << "Running on device:        " << device_ID
                                            << " (" << deviceProp.name << ")"
                                            << std::endl
              << std::endl;

    SphereType high = SphereType(1.0f, 1.0f, 1.0f, 0.1f);
    SphereType low = SphereType(-1.0f, -1.0f, -1.0f, 0.0f);

    thrust::device_vector<SphereType> d_spheres(N);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      d_spheres.begin(),
                      random_sphere_functor<SphereType>(low, high));
    grace::CudaBvh d_tree(N, max_per_leaf);
    build_tree(d_spheres, d_tree);

    grace::HostBvh h_tree(N, max_per_leaf);
    d_tree.to_host(h_tree);
    const int N_leaves = h_tree.num_leaves();
    const int N_nodes = N_leaves - 1;

    thrust::host_vector<int> parent_flags(N_nodes);
    thrust::host_vector<int> child_flags(N_nodes + N_leaves);
    thrust::host_vector<int> particle_flags(N);

    typedef grace::detail::Bvh_ref<grace::HostBvh> ref_type;
    ref_type bvh_ref(h_tree);
    for (size_t ni = 0; ni < N_nodes; ++ni)
    {
        ref_type::node_type node = bvh_ref.nodes()[ni];
        int l = node.left_child();
        int r = node.right_child();

        parent_flags[ni] += 1;
        child_flags[l] += 1;
        child_flags[r] += 1;
    }

    for (size_t li = 0; li < N_leaves; ++li)
    {
        ref_type::leaf_type leaf = bvh_ref.leaves()[li];
        int first = leaf.first_primitive();
        int size = leaf.size();

        for (int pi = first; pi < first + size; ++pi)
        {
            particle_flags[pi] += 1;
        }
    }

    size_t failures = 0;

    for (size_t ni = 0; ni < parent_flags.size(); ++ni)
    {
        int flag = parent_flags[ni];
        if (flag != 1) {
            std::cout << "Error: parent count @ " << ni << " = " << flag
                      << std::endl;
            failures += 1;
        }
    }

    for (size_t ci = 0; ci < child_flags.size(); ++ci)
    {
        int flag = child_flags[ci];

        if (ci == h_tree.root_index() && flag != 0)
        {
            std::cout << "Error: child count @ root " << ci << " = " << flag
                      << std::endl;
            failures += 1;
        }

        if (ci != h_tree.root_index() && flag != 1) {
            std::cout << "Error: child count @ " << ci << " = " << flag
                      << std::endl;
            failures += 1;
        }
    }

    for (size_t pi = 0; pi < N; ++pi)
    {
        int flag = particle_flags[pi];
        if (flag != 1) {
            std::cout << "Error: particle count @ " << pi << " = " << flag
                      << std::endl;
            failures += 1;
        }
    }

    if (failures == 0)
    {
        std::cout << "PASSED" << std::endl;
        return EXIT_SUCCESS;
    }
    else
    {
        std::cout << "FAILED" << std::endl;
        return EXIT_FAILURE;
    }
}
