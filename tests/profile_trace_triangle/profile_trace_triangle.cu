#include "hip/hip_runtime.h"
// Due to a bug in thrust, this must appear before thrust/sort.h
// The simplest solution is to put it here, despite already being included in
// all of the includes which require it.
// See http://stackoverflow.com/questions/23352122
#include <hiprand/hiprand_kernel.h>

#include "triangle.cuh"
#include "tris_tree.cuh"
#include "tris_trace.cuh"

#include "grace/cuda/nodes.h"
#include "grace/cuda/generate_rays.cuh"
#include "grace/aabb.h"
#include "grace/ray.h"
#include "grace/vector.h"
#include "helper/cuda_timer.cuh"

#include <thrust/device_vector.h>

#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>

int main(int argc, char* argv[])
{
    hipDeviceProp_t deviceProp;
    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(3);

    size_t N_rays = 512 * 512; // = 262,144
    int max_per_leaf = 32;
    std::string fname = "../data/ply/dragon_recon/dragon_vrip.ply";
    int N_iter = 10;
    unsigned int device_ID = 0;

    if (argc > 1) {
        N_rays = 32 * (size_t)std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        max_per_leaf = (int)std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        fname = std::string(argv[3]);
    }
    if (argc > 4) {
        N_iter = (int)std::strtol(argv[4], NULL, 10);
    }
    if (argc > 5) {
        device_ID = (unsigned int)std::strtol(argv[5], NULL, 10);
    }

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);

    size_t N_per_side = std::floor(std::pow(N_rays, 0.500001));
    // N_rays must be a multiple of 32.
    N_per_side = ((N_per_side + 32 - 1) / 32) * 32;
    N_rays = N_per_side * N_per_side;

{   // Device code. To ensure that hipDeviceReset() does not fail, all Thrust
    // vectors should be allocated within this block. (The closing } brace
    // causes them to be freed before we call hipDeviceReset(); if device
    // vectors are not freed, hipDeviceReset() will throw.)

    std::cout << "Input geometry file:     " << fname << std::endl;
    // Vector is resized in read_triangles().
    std::vector<PLYTriangle> ply_tris;
    thrust::device_vector<Triangle> d_tris;
    read_triangles(fname, ply_tris);
    d_tris = ply_tris;
    const size_t N = d_tris.size();

    std::cout << "Number of primitives:    " << N << std::endl
              << "Number of rays:          " << N_rays << std::endl
              << "Max primitives per leaf: " << max_per_leaf << std::endl
              << "Number of iterations:    " << N_iter << std::endl
              << "Running on device:       " << device_ID
                                             << " (" << deviceProp.name << ")"
                                             << std::endl
              << std::endl;

    grace::AABB<float> aabb;
    grace::Tree d_tree(N, max_per_leaf);
    build_tree_tris(d_tris, d_tree, &aabb);

    std::vector<grace::Vector<3, float> > camera_positions;
    grace::Vector<3, float> look_at, view_up;
    float FOVy_radians, ray_length;
    float FOVy_degrees = 50.f;
    setup_cameras(aabb, FOVy_degrees, N_per_side, N_per_side,
                  camera_positions, &look_at, &view_up,
                  &FOVy_radians, &ray_length);

    CUDATimer timer;
    double t_genray, t_closest, t_all;
    t_genray = t_closest = t_all = 0.0;
    for (int i = -1; i < N_iter; ++i)
    {
        for (int j = 0; j < camera_positions.size(); ++j)
        {
            timer.start();

            thrust::device_vector<grace::Ray> d_rays(N_rays);
            thrust::device_vector<int> d_closest_tri_idx(N_rays);
            // Don't include above memory allocations in t_genray.
            timer.split();

            pinhole_camera_rays(camera_positions[j], look_at, view_up,
                                FOVy_radians, ray_length,
                                N_per_side, N_per_side,
                                d_rays);
            if (i >= 0) t_genray += timer.split();

            trace_closest_tri(
                d_rays,
                d_tris,
                d_tree,
                d_closest_tri_idx);
            if (i >= 0) t_closest += timer.split();

            if (i >= 0) t_all += timer.elapsed();

            // Must be done in-loop for hipMemGetInfo to return relevant results.
            if (i == 0 && j == 0) {
                // Temporary memory used in tree construction is impossible to
                // (straightforwardly) compute, so below we only include the
                // 'permanently' allocated memory.
                float trace_bytes = 0.0;
                trace_bytes += d_tris.size() * sizeof(Triangle);
                trace_bytes += d_tree.leaves.size() * sizeof(int4);
                trace_bytes += d_tree.nodes.size() * sizeof(int4);
                trace_bytes += d_rays.size() * sizeof(grace::Ray);
                trace_bytes += d_closest_tri_idx.size() * sizeof(int);

                std::cout << std::endl
                          << "Total memory for closest-triangle traversal: "
                          << trace_bytes / (1024.0 * 1024.0 * 1024.0) << " GiB"
                          << std::endl;

                size_t avail, total;
                hipMemGetInfo(&avail, &total);
                std::cout << "Free memory:  " << avail / (1024.0 * 1024.0 * 1024.0)
                          << " GiB" << std::endl
                          << "Total memory: " << total / (1024.0 * 1024.0 * 1024.0)
                          << " GiB" << std::endl
                          << std::endl;
            }
        }
    }

    size_t N_trials = N_iter * camera_positions.size();
    std::cout << "Time for generating rays:             " << std::setw(8)
              << t_genray / N_trials << " ms" << std::endl
              << "Time for closest-triangle traversal:  " << std::setw(8)
              << t_closest / N_trials << " ms" << std::endl
              << "Time for total (inc. memory ops):     " << std::setw(8)
              << t_all / N_trials << " ms" << std::endl
              << std::endl;

} // End device code.

    // Exit cleanly to ensure full profiler (nvprof/nvvp) trace.
    hipDeviceReset();
    return EXIT_SUCCESS;
}
