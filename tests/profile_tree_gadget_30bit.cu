#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "../nodes.h"
#include "../utils.cuh"
#include "../device/build_functors.cuh"
#include "../kernels/albvh.cuh"
#include "../kernels/build_sph.cuh"

int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;
    std::ifstream infile;
    std::string infile_name;

    std::cout.setf(std::ios::fixed, std::ios::floatfield);
    std::cout.precision(3);


    /* Initialize run parameters. */

    unsigned int device_ID = 0;
    unsigned int max_per_leaf = 32;
    unsigned int N_iter = 100;
    infile_name = "Data_025";

    if (argc > 1) {
        device_ID = (unsigned int) std::strtol(argv[1], NULL, 10);
    }
    if (argc > 2) {
        max_per_leaf = (unsigned int) std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        N_iter = (unsigned int) std::strtol(argv[3], NULL, 10);
    }

    /* Output run parameters and device properties to console. */

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);

    std::cout << "Device " << device_ID
                    << ":                   " << deviceProp.name << std::endl;
    std::cout << "MORTON_THREADS_PER_BLOCK:   "
              << grace::MORTON_THREADS_PER_BLOCK << std::endl;
    std::cout << "BUILD_THREADS_PER_BLOCK:    "
              << grace::BUILD_THREADS_PER_BLOCK << std::endl;
    std::cout << "MAX_BLOCKS:                 "
              << grace::MAX_BLOCKS << std::endl;
    std::cout << "Max particles per leaf:     " << max_per_leaf << std::endl;
    std::cout << "Iterations per tree:        " << N_iter << std::endl;
    std::cout << "Gadget data file name:      " << infile_name << std::endl;
    std::cout << std::endl << std::endl;


    /* Read in Gadget data. */

    // Arrays are resized in read_gadget_gas()
    thrust::host_vector<float4> h_spheres_xyzr(1);
    thrust::host_vector<unsigned int> h_gadget_IDs(1);
    thrust::host_vector<float> h_masses(1);
    thrust::host_vector<float> h_rho(1);

    infile.open(infile_name.c_str(), std::ios::binary);
    grace::read_gadget_gas(infile, h_spheres_xyzr,
                                   h_gadget_IDs,
                                   h_masses,
                                   h_rho);
    infile.close();

    size_t N = h_spheres_xyzr.size();

    // Gadget IDs, masses and densities unused.
    h_gadget_IDs.clear(); h_gadget_IDs.shrink_to_fit();
    h_masses.clear(); h_masses.shrink_to_fit();
    h_rho.clear(); h_rho.shrink_to_fit();

    thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;


    /* Profile the tree constructed from Gadget data. */

    hipEvent_t part_start, part_stop;
    hipEvent_t tot_start, tot_stop;
    float part_elapsed;
    double all_tot, morton_tot, sort_tot;
    double deltas_tot, leaves_tot, leaf_deltas_tot, nodes_tot;
    hipEventCreate(&part_start);
    hipEventCreate(&part_stop);
    hipEventCreate(&tot_start);
    hipEventCreate(&tot_stop);

    for (int i=0; i<N_iter; i++) {
        hipEventRecord(tot_start);

        thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;
        thrust::device_vector<grace::uinteger32> d_keys(N);

        hipEventRecord(part_start);
        grace::morton_keys_sph(d_spheres_xyzr, d_keys);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        morton_tot += part_elapsed;

        hipEventRecord(part_start);
        thrust::sort_by_key(d_keys.begin(), d_keys.end(),
                            d_spheres_xyzr.begin());
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        sort_tot += part_elapsed;

        thrust::device_vector<float> d_deltas(N+1);

        hipEventRecord(part_start);
        grace::euclidean_deltas_sph(d_spheres_xyzr, d_deltas);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        deltas_tot += part_elapsed;

        grace::Tree d_tree(N, max_per_leaf);
        thrust::device_vector<int2> d_tmp_nodes(N - 1);

        hipEventRecord(part_start);
        grace::ALBVH::build_leaves(
            d_tmp_nodes,
            d_tree.leaves,
            d_tree.max_per_leaf,
            thrust::raw_pointer_cast(d_deltas.data()),
            thrust::less<float>());
        grace::ALBVH::remove_empty_leaves(d_tree);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        leaves_tot += part_elapsed;

        const size_t n_new_leaves = d_tree.leaves.size();
        thrust::device_vector<float> d_new_deltas(n_new_leaves + 1);

        hipEventRecord(part_start);
        grace::ALBVH::copy_leaf_deltas(
            d_tree.leaves,
            thrust::raw_pointer_cast(d_deltas.data()),
            thrust::raw_pointer_cast(d_new_deltas.data()));
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        leaf_deltas_tot += part_elapsed;

        hipEventRecord(part_start);
        grace::ALBVH::build_nodes(
            d_tree,
            thrust::raw_pointer_cast(d_spheres_xyzr.data()),
            thrust::raw_pointer_cast(d_new_deltas.data()),
            thrust::less<float>(),
            grace::AABB_sphere());
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        nodes_tot += part_elapsed;

        hipEventRecord(tot_stop);
        hipEventSynchronize(tot_stop);
        hipEventElapsedTime(&part_elapsed, tot_start, tot_stop);
        all_tot += part_elapsed;
    }

    std::cout << "Will generate a tree from " << N << " SPH particles."
              << std::endl;
    std::cout << std::endl;

    std::cout << "Time for Morton key generation:    ";
    std::cout.width(7);
    std::cout << morton_tot/N_iter << " ms." << std::endl;

    std::cout << "Time for sort-by-key:              ";
    std::cout.width(7);
    std::cout << sort_tot/N_iter << " ms." << std::endl;

    std::cout << "Time for computing deltas:         ";
    std::cout.width(7);
    std::cout << deltas_tot/N_iter << " ms." << std::endl;

    std::cout << "Time for building leaves:          ";
    std::cout.width(7);
    std::cout << leaves_tot/N_iter << " ms." << std::endl;

    std::cout << "Time for computing leaf deltas:    ";
    std::cout.width(7);
    std::cout << leaf_deltas_tot/N_iter << " ms." << std::endl;

    std::cout << "Time for building nodes:           ";
    std::cout.width(7);
    std::cout << nodes_tot/N_iter << " ms." << std::endl;

    std::cout << "Time for total (inc. memory ops):  ";
    std::cout.width(7);
    std::cout << all_tot/N_iter << " ms." << std::endl;
    std::cout << std::endl << std::endl;
}
