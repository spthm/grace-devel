#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>

rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

__device__ float3 boxnormal(float t)
{
    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
    float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
    return pos-neg;
}

RT_PROGRAM void intersect(int)
{
    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    float tmin = fmaxf( near );
    float tmax = fminf( far );

    if(tmin <= tmax) 
    {
	bool check_second = true;
	if( rtPotentialIntersection( tmin ) ) 
	{
	    texcoord = make_float3( 0.0f );
	    geometric_normal = shading_normal = boxnormal( tmin );
	    if(rtReportIntersection(0))
		check_second = false;
	} 
	if(check_second) 
	{
	    if( rtPotentialIntersection( tmax ) ) 
	    {
		texcoord = make_float3( 0.0f );
		geometric_normal = shading_normal = boxnormal( tmax );
		rtReportIntersection(0);
	    }
	}
    }
}

RT_PROGRAM void bounds (int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set(boxmin, boxmax);
}
