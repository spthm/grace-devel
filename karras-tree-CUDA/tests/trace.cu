#include "hip/hip_runtime.h"
#include <cmath>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "utils.cuh"
#include "../types.h"
#include "../nodes.h"
#include "../ray.h"
#include "../kernels/morton.cuh"
#include "../kernels/bintree_build.cuh"
#include "../kernels/bintree_trace.cuh"

int main(int argc, char* argv[])
{

    unsigned int N = 1000000;
    unsigned int N_rays = 100000;
    // Do we save the input and output data?
    bool save_data = false;

    if (argc > 3) {
        if (strcmp("save", argv[3]) == 0)
            save_data = true;
    }
    if (argc > 2) {
        N_rays = (unsigned int) std::strtol(argv[2], NULL, 10);
    }
    if (argc > 1) {
        N = (unsigned int) std::strtol(argv[1], NULL, 10);
    }

    std::cout << "Generating " << N << " random points and " << N_rays
              << " random rays." << std::endl;
    if (save_data)
        std::cout << "Will save sphere, ray and hit data." << std::endl;
    std::cout << std::endl;
{

    // Generate N random positions and radii, i.e. 4N random floats in [0,1).
    thrust::device_vector<float4> d_spheres_xyzr(N);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N),
                      d_spheres_xyzr.begin(),
                      grace::random_float4_functor(0.1f) );

    // Set the centre-containing AABBs.
    float4 bot = make_float4(0.f, 0.f, 0.f, 0.f);
    float4 top = make_float4(1.f, 1.f, 1.f, 0.f);

    // Sort the positions by their keys and save the sorted keys.
    thrust::device_vector<grace::uinteger32> d_keys(N);
    grace::morton_keys(d_spheres_xyzr, d_keys, bot, top);

    thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_spheres_xyzr.begin());

    // Build the tree hierarchy from the keys.
    grace::Nodes d_nodes(N-1);
    grace::Leaves d_leaves(N);
    grace::build_nodes(d_nodes, d_leaves, d_keys);
    // Keys no longer needed.
    d_keys.clear();
    d_keys.shrink_to_fit();
    grace::find_AABBs(d_nodes, d_leaves, d_spheres_xyzr);

    // Generate the rays (emitted from box centre (.5, .5, .5) of length 2).
    thrust::host_vector<grace::Ray> h_rays(N_rays);
    thrust::host_vector<float> h_dxs(N_rays);
    thrust::host_vector<float> h_dys(N_rays);
    thrust::host_vector<float> h_dzs(N_rays);
    thrust::host_vector<grace::uinteger32> h_keys(N_rays);
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N_rays),
                      h_dxs.begin(),
                      grace::random_float_functor(0u, -1.0f, 1.0f) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N_rays),
                      h_dys.begin(),
                      grace::random_float_functor(1u, -1.0f, 1.0f) );
    thrust::transform(thrust::counting_iterator<unsigned int>(0),
                      thrust::counting_iterator<unsigned int>(N_rays),
                      h_dzs.begin(),
                      grace::random_float_functor(2u, -1.0f, 1.0f) );
    for (int i=0; i<N_rays; i++) {
        float N_dir = sqrt(h_dxs[i]*h_dxs[i] +
                           h_dys[i]*h_dys[i] +
                           h_dzs[i]*h_dzs[i]);
        h_rays[i].dx = h_dxs[i] / N_dir;
        h_rays[i].dy = h_dys[i] / N_dir;
        h_rays[i].dz = h_dzs[i] / N_dir;
        h_rays[i].ox = h_rays[i].oy = h_rays[i].oz = 0.5f;
        h_rays[i].length = 2;
        h_rays[i].dclass = 0;
        if (h_dxs[i] >= 0)
            h_rays[i].dclass += 1;
        if (h_dys[i] >= 0)
            h_rays[i].dclass += 2;
        if (h_dzs[i] >= 0)
            h_rays[i].dclass += 4;

        // morton_key(float, float, float) requires floats in (0, 1).
        h_keys[i] = grace::morton_key((h_rays[i].dx+1)/2.f,
                                      (h_rays[i].dy+1)/2.f,
                                      (h_rays[i].dz+1)/2.f);
    }
    h_dxs.clear();
    h_dxs.shrink_to_fit();
    h_dys.clear();
    h_dys.shrink_to_fit();
    h_dzs.clear();
    h_dxs.shrink_to_fit();

    // Sort rays by Morton key and trace for per-ray hit couynts.
    thrust::sort_by_key(h_keys.begin(), h_keys.end(), h_rays.begin());
    thrust::device_vector<grace::Ray> d_rays = h_rays;
    thrust::device_vector<unsigned int> d_hit_counts(N_rays);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    grace::gpu::trace_hitcount<<<28, TRACE_THREADS_PER_BLOCK>>>(
        thrust::raw_pointer_cast(d_rays.data()),
        d_rays.size(),
        thrust::raw_pointer_cast(d_hit_counts.data()),
        thrust::raw_pointer_cast(d_nodes.hierarchy.data()),
        thrust::raw_pointer_cast(d_nodes.AABB.data()),
        d_nodes.hierarchy.size(),
        thrust::raw_pointer_cast(d_spheres_xyzr.data()));
    CUDA_HANDLE_ERR( hipPeekAtLastError() );
    CUDA_HANDLE_ERR( hipDeviceSynchronize() );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    int max_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                  0u, thrust::maximum<unsigned int>());
    int min_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                  N, thrust::minimum<unsigned int>());
    float mean_hits = thrust::reduce(d_hit_counts.begin(), d_hit_counts.end(),
                                     0u, thrust::plus<unsigned int>())
                                    / float(N_rays);
    std::cout << "Time for hit-count tracing kernel: " << elapsed
              << " ms" << std::endl;
    std::cout << std::endl;
    std::cout << "Number of rays:       " << N_rays << std::endl;
    std::cout << "Number of particles:  " << N << std::endl;
    std::cout << "Mean hits:            " << mean_hits << std::endl;
    std::cout << "Max hits:             " << max_hits << std::endl;
    std::cout << "Min hits:             " << min_hits << std::endl;

    if (save_data)
    {
        std::ofstream outfile;

        outfile.setf(std::ios::fixed, std::ios::floatfield);
        outfile.precision(9);
        outfile.width(11);
        outfile.fill('0');

        thrust::host_vector<float4> h_spheres_xyzr = d_spheres_xyzr;
        outfile.open("indata/spheredata.txt");
        for (int i=0; i<N; i++) {
            outfile << h_spheres_xyzr[i].x << " " << h_spheres_xyzr[i].y << " "
                    << h_spheres_xyzr[i].z << " " << h_spheres_xyzr[i].w
                    << std::endl;
        }
        outfile.close();

        outfile.open("indata/raydata.txt");
        for (int i=0; i<N_rays; i++) {
            outfile << h_rays[i].dx << " " << h_rays[i].dy << " "
                    << h_rays[i].dz << " " << h_rays[i].ox << " "
                    << h_rays[i].oy << " " << h_rays[i].oz << " "
                    << h_rays[i].length << std::endl;
        }
        outfile.close();

        thrust::host_vector<float> h_hit_counts = d_hit_counts;
        outfile.open("outdata/hitdata.txt");
        for (int i=0; i<N_rays; i++) {
            outfile << h_hit_counts[i] << std::endl;
        }
        outfile.close();
    }

}
    // Exit cleanly to ensure full profiler trace.
    hipDeviceReset();
    return 0;
}
