#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <sstream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include "utils.cuh"
#include "../types.h"
#include "../nodes.h"
#include "../kernels/morton.cuh"
#include "../kernels/bintree_build.cuh"

int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;
    std::ofstream outfile;
    std::ifstream infile;
    std::string infile_name, outfile_name;
    std::ostringstream converter;

    outfile.setf(std::ios::fixed, std::ios::floatfield);
    outfile.precision(5);


    /* Initialize run parameters. */

    unsigned int file_num = 1;
    unsigned int device_ID = 0;
    unsigned int N_iter = 100;
    infile_name = "Data_025";

    if (argc > 1) {
        file_num = (unsigned int) std::strtol(argv[1], NULL, 10);

    }
    if (argc > 2) {
        device_ID = (unsigned int) std::strtol(argv[2], NULL, 10);
    }
    if (argc > 3) {
        N_iter = (unsigned int) std::strtol(argv[3], NULL, 10);
    }


    // Converts file number to a string.
    converter << file_num;
    outfile_name = ("profile_tree_gadget_" + converter.str() + ".log");

    std::cout << "Will profile (on device " << device_ID << " with " << N_iter
              << " iterations) a tree from Gadget file " << infile_name
              << std::endl;
    std::cout << std::endl;
    std::cout << "Saving results to " << outfile_name << std::endl;
    std::cout << std::endl;

    infile.open(infile_name.c_str(), std::ios::binary);
    grace::gadget_header header = grace::read_gadget_header(infile);
    infile.close();


    /* Write run parameters to file. */

    hipGetDeviceProperties(&deviceProp, device_ID);
    hipSetDevice(device_ID);
    // Wipe the file, if it exists.
    outfile.open(outfile_name.c_str(), std::ofstream::trunc);
    outfile << "Device " << device_ID
                    << ":                   " << deviceProp.name << std::endl;
    outfile << "MORTON_THREADS_PER_BLOCK:   " << MORTON_THREADS_PER_BLOCK
            << std::endl;
    outfile << "BUILD_THREADS_PER_BLOCK:    " << BUILD_THREADS_PER_BLOCK
            << std::endl;
    outfile << "AABB_THREADS_PER_BLOCK:     " << AABB_THREADS_PER_BLOCK
            << std::endl;
    outfile << "MAX_BLOCKS:                 " << MAX_BLOCKS << std::endl;
    outfile << "Iterations per tree:        " << N_iter << std::endl;
    outfile << "Gadget data file name:      " << infile_name << std::endl;
    outfile << std::endl << std::endl;
    outfile.close();


    std::cout << "Profiling tree from Gadget data."<< std::endl;
    unsigned int N = header.npart[0];
    std::cout << "Gadget files contains " << N << " gas particles and "
              << header.npart[1] << " dark matter particles." << std::endl;


    /* Read in Gadget data. */

    thrust::host_vector<float4> h_spheres_xyzr(N);
    thrust::host_vector<float> h_masses(N);
    thrust::host_vector<float> h_rho(N);

    std::cout << "Reading in file..." << std::endl;
    infile.open(infile_name.c_str(), std::ios::binary);
    grace::read_gadget_gas(infile, h_spheres_xyzr, h_masses, h_rho);
    infile.close();
    // Masses unused.  Free space.
    h_masses.clear(); h_masses.shrink_to_fit();
    h_rho.clear(); h_rho.shrink_to_fit();

    thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;

    // Set the tree-build AABB (contains all sphere centres).
    float min_x, max_x;
    grace::min_max_x(&min_x, &max_x, d_spheres_xyzr);

    float min_y, max_y;
    grace::min_max_y(&min_y, &max_y, d_spheres_xyzr);

    float min_z, max_z;
    grace::min_max_z(&min_z, &max_z, d_spheres_xyzr);

    float4 bot = make_float4(min_x, min_y, min_z, 0.f);
    float4 top = make_float4(max_x, max_y, max_z, 0.f);


    /* Profile the tree constructed from Gadget data. */

    hipEvent_t part_start, part_stop;
    hipEvent_t tot_start, tot_stop;
    float part_elapsed;
    double all_tot, morton_tot, sort_tot, tree_tot, aabb_tot;
    hipEventCreate(&part_start);
    hipEventCreate(&part_stop);
    hipEventCreate(&tot_start);
    hipEventCreate(&tot_stop);

    for (int i=0; i<N_iter; i++) {
        hipEventRecord(tot_start);
        // Copy pristine host-side data to GPU.
        thrust::device_vector<float4> d_spheres_xyzr = h_spheres_xyzr;

        thrust::device_vector<grace::uinteger32> d_keys(N);
        hipEventRecord(part_start);
        grace::morton_keys(d_spheres_xyzr, d_keys, bot, top);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        morton_tot += part_elapsed;

        hipEventRecord(part_start);
        thrust::sort_by_key(d_keys.begin(), d_keys.end(),
                            d_spheres_xyzr.begin());
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        sort_tot += part_elapsed;

        grace::Nodes d_nodes(N-1);
        grace::Leaves d_leaves(N);
        hipEventRecord(part_start);
        grace::build_nodes(d_nodes, d_leaves, d_keys);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        tree_tot += part_elapsed;

        hipEventRecord(part_start);
        grace::find_AABBs(d_nodes, d_leaves, d_spheres_xyzr);
        hipEventRecord(part_stop);
        hipEventSynchronize(part_stop);
        hipEventElapsedTime(&part_elapsed, part_start, part_stop);
        aabb_tot += part_elapsed;

        // Record the total time spent in the loop.
        hipEventRecord(tot_stop);
        hipEventSynchronize(tot_stop);
        hipEventElapsedTime(&part_elapsed, tot_start, tot_stop);
        all_tot += part_elapsed;
    }

    outfile.open(outfile_name.c_str(),
                 std::ofstream::out | std::ofstream::app);
    outfile << "Will generate:" << std::endl;
    outfile << "    i)  A tree from " << N << " SPH particles." << std::endl;
    outfile << std::endl;
    outfile << "Time for Morton key generation:    ";
    outfile.width(8);
    outfile << morton_tot/N_iter << " ms." << std::endl;
    outfile << "Time for sort-by-key:              ";
    outfile.width(8);
    outfile << sort_tot/N_iter << " ms." << std::endl;
    outfile << "Time for hierarchy generation:     ";
    outfile.width(8);
    outfile << tree_tot/N_iter << " ms." << std::endl;
    outfile << "Time for calculating AABBs:        ";
    outfile.width(8);
    outfile << aabb_tot/N_iter << " ms." << std::endl;
    outfile << "Time for total (inc. memory ops): ";
    outfile.width(8);
    outfile << all_tot/N_iter << " ms." << std::endl;
    outfile << std::endl << std::endl;
    outfile.close();
}
